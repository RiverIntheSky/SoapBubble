# include "../include/KaminoTimer.cuh"

KaminoTimer::KaminoTimer() : timeElapsed(0.0f)
{
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
}
KaminoTimer::~KaminoTimer()
{
    hipEventDestroy(start);
    hipEventDestroy(stop);
}
void KaminoTimer::startTimer()
{
    hipEventRecord(start, 0);
}
float KaminoTimer::stopTimer()
{
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    checkCudaErrors(hipEventElapsedTime(&timeElapsed, start, stop));
    return timeElapsed;
}