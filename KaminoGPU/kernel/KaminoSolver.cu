#include "hip/hip_runtime.h"
# include "../include/KaminoSolver.cuh"
# include "../opencv_headers/opencv2/opencv.hpp"
# include "../include/KaminoTimer.cuh"
# include "../include/tinyexr.h"

// CONSTRUCTOR / DESTRUCTOR >>>>>>>>>>

const int fftRank = 1;
static const int m = 3;
static __constant__ size_t nPhiGlobal;
static __constant__ size_t nThetaGlobal;
static __constant__ fReal gridLenGlobal;
static __constant__ int Cols;
static __constant__ int Rows;
static __constant__ int Ratio;


__device__ fReal sampleThickness(fReal* input, fReal phiRawId, fReal thetaRawId) {
    fReal phi = phiRawId - centeredPhiOffset;
    fReal theta = thetaRawId - centeredThetaOffset;
    // Phi and Theta are now shifted back to origin

    // if (thetaRawId > nThetaGlobal)
    // 	printf("theta %f\n", theta);
    // TODO change for particles
    size_t nPhi = static_cast<size_t>(Cols);
    bool isFlippedPole = validateCoord(phi, theta, nPhi);

    int phiIndex = static_cast<int>(floorf(phi));
    int thetaIndex = static_cast<int>(floorf(theta));
    fReal alphaPhi = phi - static_cast<fReal>(phiIndex);
    fReal alphaTheta = theta - static_cast<fReal>(thetaIndex);

    if (thetaIndex == 0 && isFlippedPole) {
	size_t phiLower = phiIndex % Cols;
	size_t phiHigher = (phiLower + 1) % Cols;
	fReal higherBelt = kaminoLerp(input[phiLower + Cols * thetaIndex],
				      input[phiHigher + Cols * thetaIndex], alphaPhi);

	phiLower = (phiLower + Cols / 2) % Cols;
	phiHigher = (phiHigher + Cols / 2) % Cols;
	fReal lowerBelt = kaminoLerp(input[phiLower + Cols * thetaIndex],
				     input[phiHigher + Cols * thetaIndex], alphaPhi);

	fReal lerped = kaminoLerp(lowerBelt, higherBelt, alphaTheta);
	return lerped;
    }
    
    if (isFlippedPole) {
	thetaIndex -= 1;
    }
    
    if (thetaIndex == Rows - 1) {
	size_t phiLower = phiIndex % Cols;
	size_t phiHigher = (phiLower + 1) % Cols;
	fReal lowerBelt = kaminoLerp(input[phiLower + Cols * thetaIndex],
				     input[phiHigher + Cols * thetaIndex], alphaPhi);

	phiLower = (phiLower + Cols / 2) % Cols;
	phiHigher = (phiHigher + Cols / 2) % Cols;
	fReal higherBelt = kaminoLerp(input[phiLower + Cols * thetaIndex],
				      input[phiHigher + Cols * thetaIndex], alphaPhi);

	fReal lerped = kaminoLerp(lowerBelt, higherBelt, alphaTheta);
	return lerped;
    }

    size_t phiLower = phiIndex % Cols;
    size_t phiHigher = (phiLower + 1) % Cols;
    size_t thetaLower = thetaIndex;
    size_t thetaHigher = thetaIndex + 1;

    fReal lowerBelt = kaminoLerp(input[phiLower + Cols * thetaLower],
				 input[phiHigher + Cols * thetaLower], alphaPhi);
    fReal higherBelt = kaminoLerp(input[phiLower + Cols * thetaHigher],
				  input[phiHigher + Cols * thetaHigher], alphaPhi);

    fReal lerped = kaminoLerp(lowerBelt, higherBelt, alphaTheta);
    return lerped;
}


__global__ void initParticleValues(fReal* particleVal, fReal* particleCoord, fReal* thickness) {
    int particleId = blockIdx.x * blockDim.x + threadIdx.x;
    fReal phiId = particleCoord[2 * particleId] * Ratio;
    fReal thetaId = particleCoord[2 * particleId + 1] * Ratio;

    particleVal[particleId] = sampleThickness(thickness, phiId, thetaId);
}


__global__ void initLinearSystem(int* row_ptr, int* col_ind, int* row_ptrm, int* col_indm, float* valm) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    int idx5 = 5 * idx;
    row_ptr[idx] = idx5;
    row_ptrm[idx] = idx;

    int n = nPhiGlobal * nThetaGlobal;
    
    if (idx < nPhiGlobal * nThetaGlobal) {
	// up
	if (idx < nPhiGlobal) { // first row
	    col_ind[idx5] = (idx + nThetaGlobal) % nPhiGlobal;
	} else {
	    col_ind[idx5] = idx - nPhiGlobal;
	}
	
	// left
	col_ind[idx5 + 1] = (idx % nPhiGlobal) == 0 ? idx + nPhiGlobal - 1 : idx - 1;
	
	// center
	col_ind[idx5 + 2] = idx;
	
	// right
	col_ind[idx5 + 3] = (idx % nPhiGlobal) == (nPhiGlobal - 1) ? idx - nPhiGlobal + 1 : idx + 1;
	
	// down
	if (idx >= (nThetaGlobal - 1) * nPhiGlobal + nThetaGlobal) {
	    // last half of the last row
	    col_ind[idx5 + 4] = idx - nThetaGlobal;
	} else if (idx >= (nThetaGlobal - 1) * nPhiGlobal) {
	    // first half of the last row
	    col_ind[idx5 + 4] = idx + nThetaGlobal;
	} else {
	    col_ind[idx5 + 4] = idx + nPhiGlobal;
	}
	col_indm[idx] = idx;
	valm[idx] = 1.f; /* identity matrix */
    }
}


KaminoSolver::KaminoSolver(size_t nPhi, size_t nTheta, fReal radius, fReal frameDuration,
			   fReal A, int B, int C, int D, int E, fReal H, int device) :
    nPhi(nPhi), nTheta(nTheta), radius(radius), invRadius(1.0/radius), gridLen(M_2PI / nPhi), invGridLen(1.0 / gridLen), frameDuration(frameDuration),
    timeStep(0.0), timeElapsed(0.0), advectionTime(0.0), bodyforceTime(0.0), CGTime(0.0),
    A(A), B(B), C(C), D(D), E(E), H(H), epsilon(H/radius), N(nPhi*nTheta), nz(5*N)
{
    /// FIXME: Should we detect and use device 0?
    /// Replace it later with functions from hip/hip_runtime_api.h!
    checkCudaErrors(hipSetDevice(device));

    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, device));
    this->nThreadxMax = min(deviceProp.maxThreadsDim[0], 128);
    
    checkCudaErrors(hipMalloc((void **)&gpuUFourier,
			       sizeof(ComplexFourier) * N));
    checkCudaErrors(hipMalloc((void **)&gpuUReal,
			       sizeof(fReal) * N));
    checkCudaErrors(hipMalloc((void **)&gpuUImag,
			       sizeof(fReal) * N));

    checkCudaErrors(hipMalloc((void **)&gpuFFourier,
			       sizeof(ComplexFourier) * N));
    checkCudaErrors(hipMalloc((void **)&gpuFReal,
			       sizeof(fReal) * N));
    checkCudaErrors(hipMalloc((void **)&gpuFImag,
			       sizeof(fReal) * N));
    checkCudaErrors(hipMalloc((void**)&gpuFZeroComponent,
			       sizeof(fReal) * nTheta));

    checkCudaErrors(hipMalloc((void **)(&div),
			       sizeof(fReal) * N));
    checkCudaErrors(hipMalloc((void **)(&weight),
			       sizeof(float2) * N));
    checkCudaErrors(hipMalloc((void **)(&row_ptr),
			       sizeof(int) * (N + 1)));
    checkCudaErrors(hipMalloc((void **)(&col_ind),
			       sizeof(int) * nz));
    checkCudaErrors(hipMalloc((void **)(&rhs),
			       sizeof(float) * N));
    checkCudaErrors(hipMalloc((void **)(&val),
			       sizeof(float) * nz));
    checkCudaErrors(hipMalloc((void **)(&row_ptrm),
			       sizeof(int) * (N + 1)));
    checkCudaErrors(hipMalloc((void **)(&col_indm),
			       sizeof(int) * N));
    checkCudaErrors(hipMalloc((void **)(&valm),
			       sizeof(float) * N));
    
    checkCudaErrors(hipMalloc((void **)(&gpuA),
			       sizeof(fReal) * N));
    checkCudaErrors(hipMalloc((void **)(&gpuB),
			       sizeof(fReal) * N));
    checkCudaErrors(hipMalloc((void **)(&gpuC),
			       sizeof(fReal) * N));

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(nPhiGlobal), &(this->nPhi), sizeof(size_t)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(nThetaGlobal), &(this->nTheta), sizeof(size_t)));

    this->velPhi = new KaminoQuantity("velPhi", nPhi, nTheta,
				      vPhiPhiOffset, vPhiThetaOffset);
    this->velTheta = new KaminoQuantity("velTheta", nPhi, nTheta - 1,
					vThetaPhiOffset, vThetaThetaOffset);
    this->pressure = new KaminoQuantity("p", nPhi, nTheta,
					centeredPhiOffset, centeredThetaOffset);
    this->density = new KaminoQuantity("density", nPhi, nTheta,
				       centeredPhiOffset, centeredThetaOffset);
    this->thickness = new KaminoQuantity("eta", nPhi, nTheta,
					 centeredPhiOffset, centeredThetaOffset);
    this->bulkConcentration = new KaminoQuantity("c", nPhi, nTheta,
						 centeredPhiOffset, centeredThetaOffset);
    this->surfConcentration = new KaminoQuantity("gamma", nPhi, nTheta,
						 centeredPhiOffset, centeredThetaOffset);
			   				   

    initWithConst(this->velPhi, 0.0);
    // initialize_velocity();
    initWithConst(this->velTheta, 0.0);
    initWithConst(this->thickness, 1.0);
    initWithConst(this->surfConcentration, 1.0);

    dim3 gridLayout;
    dim3 blockLayout;
			        
    determineLayout(gridLayout, blockLayout, 1, N + 1);
    initLinearSystem<<<gridLayout, blockLayout>>>(row_ptr, col_ind, row_ptrm, col_indm, valm);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    
    // printGPUarraytoMATLAB<int>("test/row_ptr.txt", row_ptr, N + 1, 1, 1);
    // printGPUarraytoMATLAB<int>("test/col_ind.txt", col_ind, N, 5, 5);
    
    // cuSPARSE and cuBLAS
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));
    CHECK_CUSPARSE(hipsparseCreate(&cusparseHandle));

    // Device memory management
    CHECK_CUDA(hipMalloc((void**)&d_x, N * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_r, N * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_p, N * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_omega, N * sizeof(float)));   
	
    // Create preconditioner and dense vectors
    CHECK_CUSPARSE(hipsparseCreateCsr(&matM, N, N, N, row_ptrm, col_indm,
    				     valm, HIPSPARSE_INDEX_32I,
    				     HIPSPARSE_INDEX_32I,
    				     HIPSPARSE_INDEX_BASE_ZERO,
    				     HIP_R_32F));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecX, N, d_x, HIP_R_32F));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecR, N, d_r, HIP_R_32F));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecP, N, d_p, HIP_R_32F));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecO, N, d_omega, HIP_R_32F));
    
    int sigLenArr[1];
    sigLenArr[0] = nPhi;
    checkCudaErrors((hipError_t)hipfftPlanMany(&kaminoPlan, fftRank, sigLenArr,
					       NULL, 1, nPhi,
					       NULL, 1, nPhi,
					       HIPFFT_C2C, nTheta));
}


KaminoSolver::~KaminoSolver()
{
    checkCudaErrors(hipFree(gpuUFourier));
    checkCudaErrors(hipFree(gpuUReal));
    checkCudaErrors(hipFree(gpuUImag));

    checkCudaErrors(hipFree(gpuFFourier));
    checkCudaErrors(hipFree(gpuFReal));
    checkCudaErrors(hipFree(gpuFImag));
    checkCudaErrors(hipFree(gpuFZeroComponent));

    checkCudaErrors(hipFree(div));
    checkCudaErrors(hipFree(weight));

    checkCudaErrors(hipFree(thicknessFull));
    checkCudaErrors(hipFree(weightFull));
		    
    checkCudaErrors(hipFree(gpuA));
    checkCudaErrors(hipFree(gpuB));
    checkCudaErrors(hipFree(gpuC));

    CHECK_CUSPARSE(hipsparseDestroySpMat(matM));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecX));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecR));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecP));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecO));
    CHECK_CUBLAS(hipblasDestroy(cublasHandle));
    CHECK_CUSPARSE(hipsparseDestroy(cusparseHandle));

    CHECK_CUDA(hipFree(d_r));
    CHECK_CUDA(hipFree(d_x));
    CHECK_CUDA(hipFree(d_p));
    CHECK_CUDA(hipFree(d_omega));
    CHECK_CUDA(hipFree(row_ptr));
    CHECK_CUDA(hipFree(col_ind));
    CHECK_CUDA(hipFree(rhs));
    CHECK_CUDA(hipFree(val));
    CHECK_CUDA(hipFree(row_ptrm));
    CHECK_CUDA(hipFree(col_indm));
    CHECK_CUDA(hipFree(valm));

    delete this->velPhi;
    delete this->velTheta;
    delete this->pressure;
    delete this->density;
    delete this->thickness;
    delete this->bulkConcentration;
    delete this->surfConcentration;
    delete this->thicknessFullCPU;

# ifdef WRITE_PARTICLES
    delete this->particles;
# endif

    checkCudaErrors(hipDeviceReset());

# ifdef PERFORMANCE_BENCHMARK
    float totalTimeUsed = this->advectionTime + this->bodyforceTime;
    std::cout << "Total time used for advection : " << this->advectionTime << std::endl;
    std::cout << "Total time used for body force : " << this->bodyforceTime << std::endl;
    std::cout << "Percentage of advection : " << advectionTime / totalTimeUsed * 100.0f << "%" << std::endl;
    std::cout << "Percentage of bodyforce : " << bodyforceTime / totalTimeUsed * 100.0f << "%" << std::endl;
    std::cout << "Percentage of CG / bodyforce : " << CGTime / bodyforceTime * 100.0f << "%" << std::endl;
# endif
}

void KaminoSolver::copyVelocity2GPU()
{
    velPhi->copyToGPU();
    velTheta->copyToGPU();
}

void KaminoSolver::copyDensity2GPU()
{
    density->copyToGPU();
}


void KaminoSolver::determineLayout(dim3& gridLayout, dim3& blockLayout,
				   size_t nTheta_row, size_t nPhi_col)
{
    if (nPhi_col <= this->nThreadxMax)
	{
	    gridLayout = dim3(nTheta_row);
	    blockLayout = dim3(nPhi_col);
	}
    else
	{
	    int splitVal = (nPhi_col + nThreadxMax - 1) / nThreadxMax;

	    gridLayout = dim3(nTheta_row * splitVal);
	    blockLayout = dim3(nThreadxMax);
	}
}


void KaminoSolver::copyToCPU(KaminoQuantity* quantity, fReal* cpubuffer) {
    checkCudaErrors(hipMemcpy2D((void*)cpubuffer, quantity->getNPhi() * sizeof(fReal),
				 (void*)quantity->getGPUThisStep(),
				 quantity->getThisStepPitchInElements() * sizeof(fReal),
				 quantity->getNPhi() * sizeof(fReal), quantity->getNTheta(),
				 hipMemcpyDeviceToHost));
}


fReal KaminoSolver::maxAbsDifference(const fReal* A, const fReal* B, const size_t& size) {
    fReal res = 0.f;
    for (int i = 0; i < size; i++) {
	fReal diff = std::abs(A[i] - B[i]);
	if (diff > res)
	    res = diff;
    }
    return res;
}


void KaminoSolver::adjustStepSize(fReal& dt, const fReal& U, const fReal& epsilon) {
    copyVelocityBack2CPU();
    thickness->copyBackToCPU();
    surfConcentration->copyBackToCPU();

    size_t sizePhiAndCentered = velPhi->getNPhi() * velPhi->getNTheta();
    size_t sizeTheta = velTheta->getNPhi() * velTheta->getNTheta();
    
    fReal* uSmall = new fReal[sizePhiAndCentered];
    fReal* vSmall = new fReal[sizeTheta];
    fReal* deltaSmall = new fReal[sizePhiAndCentered];
    fReal* gammaSmall = new fReal[sizePhiAndCentered];

    fReal* uLarge = new fReal[sizePhiAndCentered];
    fReal* vLarge = new fReal[sizeTheta];
    fReal* deltaLarge = new fReal[sizePhiAndCentered];
    fReal* gammaLarge = new fReal[sizePhiAndCentered];

    fReal optTimeStep = dt;
    int loop = 0;
    while (true) {

	// m small steps
	for (int i = 0; i < m; i++) {
	    stepForward(dt);
	}

	// store results in cpu;
	copyToCPU(velPhi, uSmall);
	copyToCPU(velTheta, vSmall);
	copyToCPU(thickness, deltaSmall);
	copyToCPU(surfConcentration, gammaSmall);
    
	// reload values before the small steps
	copyVelocity2GPU();
	thickness->copyToGPU();
	surfConcentration->copyToGPU();

	// a large step
	stepForward(dt * m);

	// store results in cpu;
	copyToCPU(velPhi, uLarge);
	copyToCPU(velTheta, vLarge);
	copyToCPU(thickness, deltaLarge);
	copyToCPU(surfConcentration, gammaLarge);
    
	// reload values before the large step
	copyVelocity2GPU();
	thickness->copyToGPU();
	surfConcentration->copyToGPU();

	fReal maxError = std::max(maxAbsDifference(uSmall, uLarge, sizePhiAndCentered),
				  std::max(maxAbsDifference(vSmall, vLarge, sizeTheta),
					   std::max(maxAbsDifference(deltaSmall, deltaLarge, sizePhiAndCentered),
						    maxAbsDifference(gammaSmall, gammaLarge, sizePhiAndCentered))));

	// optimal step size
	std::cout << "maxError " << maxError << std::endl;
	// std::cout << "epsilon " << epsilon << std::endl;
	// std::cout << "dt " << dt << std::endl;
	optTimeStep = dt * std::sqrt(epsilon* (m*m-1)/maxError);
	std::cout << "opt " << optTimeStep << std::endl;
		
	//optTimeStep = max(optTimeStep, 1e-7);

	if ((optTimeStep > 2 * dt || dt > 2 * optTimeStep) && loop < 2) {
	    loop++;
	    dt = optTimeStep;	
	} else {
	    dt = optTimeStep;
	    break;
	}
    }
    
    delete[] uSmall;
    delete[] vSmall;
    delete[] deltaSmall;
    delete[] gammaSmall;
    delete[] uLarge;
    delete[] vLarge;
    delete[] deltaLarge;
    delete[] gammaLarge;
}


void KaminoSolver::stepForward(fReal timeStep) {
    advection(timeStep);
    bodyforce();
}


void KaminoSolver::stepForward() {
    this->timeStep = timeStep;
    
# ifdef PERFORMANCE_BENCHMARK
    KaminoTimer timer;
    timer.startTimer();
# endif

    advection();
# ifdef PERFORMANCE_BENCHMARK
    this->advectionTime += timer.stopTimer() * 0.001f;
    timer.startTimer();
# endif
    bodyforce();
# ifdef PERFORMANCE_BENCHMARK
    this->bodyforceTime += timer.stopTimer() * 0.001f;
# endif
    this->timeElapsed += this->timeStep;
}


void KaminoSolver::swapVelocityBuffers()
{
    this->velPhi->swapGPUBuffer();
    this->velTheta->swapGPUBuffer();
}


void KaminoSolver::copyVelocityBack2CPU()
{
    this->velPhi->copyBackToCPU();
    this->velTheta->copyBackToCPU();
}


void KaminoSolver::copyDensityBack2CPU()
{
    this->density->copyBackToCPU();
}


void KaminoSolver::initWithConst(KaminoQuantity* attrib, fReal val)
{
    for (size_t i = 0; i < attrib->getNPhi(); ++i) {
	for (size_t j = 0; j < attrib->getNTheta(); ++j) {
		attrib->setCPUValueAt(i, j, val);
	    }
    }
    attrib->copyToGPU();
}


bool KaminoSolver::isBroken() {
    return this->broken;
}


void KaminoSolver::setBroken(bool broken) {
    this->broken = broken;
}


void KaminoSolver::initThicknessfromPic(std::string path, size_t particleDensity)
{
    if (path == "") {
	std::cout << "No thickness image provided, initialize with eta = 1." << std::endl;
    } else {
	cv::Mat image_In, image_Flipped;
	image_In = cv::imread(path, cv::IMREAD_UNCHANGED);
	if (!image_In.data) {
	    std::cout << "No thickness image provided, initialize with eta = 1." << std::endl;
	} else {
	    cv::Mat image_Resized;
	    cv::flip(image_In, image_Flipped, 1);
	    cv::Size size(nPhi, nTheta);
	    cv::resize(image_Flipped, image_Resized, size);

	    for (size_t i = 0; i < nPhi; ++i) {
		for (size_t j = 0; j < nTheta; ++j) {
		    cv::Point3_<float>* p = image_Resized.ptr<cv::Point3_<float>>(j, i);
		    fReal C = p->x; // Gray Scale
		    this->thickness->setCPUValueAt(i, j, C);
		}
	    }
	    this->thickness->copyToGPU();
    
	    this->rows = image_Flipped.rows;
	    this->cols = image_Flipped.cols;
	    int ratio = this->rows / nTheta;
	    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(Cols), &(this->cols), sizeof(int)));
	    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(Rows), &(this->rows), sizeof(int)));
	    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(Ratio), &ratio, sizeof(int)));

	    thicknessFullCPU = new fReal[rows * cols];
	    checkCudaErrors(hipMalloc((void **)&thicknessFull,
				       sizeof(fReal) * rows * cols));
	    checkCudaErrors(hipMalloc((void **)(&weightFull),
				       sizeof(float2) * rows * cols));

	    for (int i = 0; i < cols; ++i) {
		for (int j = 0; j < rows; ++j) {
		    cv::Point3_<float>* p = image_Flipped.ptr<cv::Point3_<float>>(j, i);
		    fReal C = p->x; // Gray Scale
		    thicknessFullCPU[j * cols + i] = C;
		}
	    }

	    hipMemcpy(thicknessFull, thicknessFullCPU, (cols * rows) * sizeof(fReal), hipMemcpyHostToDevice);
   	}

    }    

    this->particles = new KaminoParticles(path, particleDensity, gridLen, nTheta);

    if (particleDensity > 0) {
	dim3 gridLayout;
	dim3 blockLayout;
	determineLayout(gridLayout, blockLayout, 1, this->particles->numOfParticles);
	initParticleValues<<<gridLayout, blockLayout>>>
	    (this->particles->value, this->particles->coordGPUThisStep, thicknessFull);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());	
    }
}


void KaminoSolver::initParticlesfromPic(std::string path, size_t parPerGrid)
{
    this->particles = new KaminoParticles(path, parPerGrid, gridLen, nTheta);
}


void KaminoSolver::write_image(const std::string& s, size_t width, size_t height, std::vector<float> *images) {
    const char *filename = s.c_str();
    
    EXRHeader header;
    InitEXRHeader(&header);

    EXRImage image;
    InitEXRImage(&image);

    image.num_channels = 3;

    float* image_ptr[3];
    image_ptr[0] = &(images[2].at(0)); // B
    image_ptr[1] = &(images[1].at(0)); // G
    image_ptr[2] = &(images[0].at(0)); // R

    image.images = (unsigned char**)image_ptr;
    image.width = width;
    //    image.height = nTheta;
    image.height = height;

    header.num_channels = 3;
    header.channels = (EXRChannelInfo *)malloc(sizeof(EXRChannelInfo) * header.num_channels); 
    // Must be (A)BGR order, since most of EXR viewers expect this channel order.
    strncpy(header.channels[0].name, "B", 255); header.channels[0].name[strlen("B")] = '\0';
    strncpy(header.channels[1].name, "G", 255); header.channels[1].name[strlen("G")] = '\0';
    strncpy(header.channels[2].name, "R", 255); header.channels[2].name[strlen("R")] = '\0';

    header.pixel_types = (int *)malloc(sizeof(int) * header.num_channels); 
    header.requested_pixel_types = (int *)malloc(sizeof(int) * header.num_channels);
    for (int i = 0; i < header.num_channels; i++) {
	header.pixel_types[i] = TINYEXR_PIXELTYPE_FLOAT; // pixel type of input image
	header.requested_pixel_types[i] = TINYEXR_PIXELTYPE_HALF; // pixel type of output image to be stored in .EXR
    }

    const char* err = NULL; // or nullptr in C++11 or later.
    int ret = SaveEXRImageToFile(&image, &header, filename, &err);
    if (ret != TINYEXR_SUCCESS) {
	fprintf(stderr, "Save EXR err: %s\n", err);
	FreeEXRErrorMessage(err); // free's buffer for an error message
	return;
    }
    printf("Saved exr file. [ %s ] \n", filename);

    free(header.channels);
    free(header.pixel_types);
    free(header.requested_pixel_types);
}


void KaminoSolver::write_velocity_image(const std::string& s, const int frame) {
    std::string img_string = std::to_string(frame);
    while (img_string.length() < 4) {
	img_string.insert(0, "0");
    }
    img_string.insert(0, s);
    std::string u_string = img_string;
    std::string v_string = img_string;
    img_string.append(".exr");
    u_string.append("u.txt");
    v_string.append("v.txt");

    std::ofstream ofu(u_string);
    std::ofstream ofv(v_string);

    copyVelocityBack2CPU();
    std::vector<float> images[3];
    images[0].resize(nPhi * nTheta);
    images[1].resize(nPhi * nTheta);
    images[2].resize(nPhi * nTheta);

    fReal maxu = std::numeric_limits<fReal>::min();
    fReal maxv = std::numeric_limits<fReal>::min();
    fReal minu = std::numeric_limits<fReal>::max();
    fReal minv = std::numeric_limits<fReal>::max();
    size_t maxuthetaid = 0;
    size_t maxuphiid = 0;
    size_t minuthetaid = 0;
    size_t minuphiid = 0;
    size_t maxvthetaid = 0;
    size_t maxvphiid = 0;
    size_t minvthetaid = 0;
    size_t minvphiid = 0;

    for (size_t j = 0; j < nTheta; ++j) {
	for (size_t i = 0; i < nPhi; ++i) {
	    fReal uW = velPhi->getCPUValueAt(i, j);
	    // ofu << uW << " ";
	    fReal uE;
	    fReal vN;
	    fReal vS;
	    if (i != nPhi - 1) {
	        uE = velPhi->getCPUValueAt(i + 1, j);		
	    } else {
		uE = velPhi->getCPUValueAt(0, j);	
	    }
	    if (j != 0) {
		vN = velTheta->getCPUValueAt(i, j - 1);
	    } else {
		size_t oppositei = (i + nPhi/2) % nPhi;
		vN = 0.75 * velTheta->getCPUValueAt(i, j) -
		    0.25 * velTheta->getCPUValueAt(oppositei, j);
	    }
	    if (j != nTheta - 1) {
		vS = velTheta->getCPUValueAt(i, j);
		// ofv << vS << " ";
	    } else {
		size_t oppositei = (i + nPhi/2) % nPhi;
		vS = 0.75 * velTheta->getCPUValueAt(i, j - 1) -
		    0.25 * velTheta->getCPUValueAt(oppositei, j - 1);
	    }
	    fReal u = 0.5 * (uW + uE);
	    ofu << u << " ";
	    fReal v = 0.5 * (vN + vS);
	    ofv << v << " ";
	    if (u > maxu) {
		maxu = u;
		maxuthetaid = j;
		maxuphiid = i;
	    }
		
	    if (u < minu) {
		minu = u;
		minuthetaid = j;
		minuphiid = i;
	    }		
	    if (v > maxv) {
		maxv = v;
		maxvthetaid = j;
		maxvphiid = i;
	    }		
	    if (v < minv) {
		minv = v;
		minvthetaid = j;
		minvphiid = i;
	    }
		
	    // std::cout << "theta " << j << " phi " << i << " u " << u << " v " << v << std::endl;
	    images[0][j*nPhi+i] = u/2+0.5; // R
	    images[1][j*nPhi+i] = v/2+0.5; // G
	    images[2][j*nPhi+i] = 0.5; // B
	}
	ofu << std::endl;
	ofv << std::endl;
    }

    std::cout << "max u = " << maxu << " theta " << maxuthetaid << " phi " << maxuphiid << std::endl;
    std::cout << "min u = " << minu << " theta " << minuthetaid << " phi " << minuphiid << std::endl;
    std::cout << "max v = " << maxv << " theta " << maxvthetaid << " phi " << maxvphiid << std::endl;
    std::cout << "min v = " << minv << " theta " << minvthetaid << " phi " << minvphiid << std::endl;

    write_image(img_string, nPhi, nTheta, images);
}


void KaminoSolver::write_concentration_image(const std::string& s, const int frame) {
    std::string img_string = std::to_string(frame);
    while (img_string.length() < 4) {
	img_string.insert(0, "0");
    }
    img_string.insert(0, s);
    std::string mat_string = img_string;
    img_string.append(".exr");
    mat_string.append(".txt");

    std::ofstream of(mat_string);

    surfConcentration->copyBackToCPU();
    std::vector<float> images[3];
    images[0].resize(nPhi * nTheta);
    images[1].resize(nPhi * nTheta);
    images[2].resize(nPhi * nTheta);

    for (size_t j = 0; j < nTheta; ++j) {
	for (size_t i = 0; i < nPhi; ++i) {
	    fReal con = surfConcentration->getCPUValueAt(i, j);
	  
	    // std::cout << "theta " << j << " phi " << i << " u " << u << " v " << v << std::endl;
	    images[0][j*nPhi+i] = con*0.2;
	    images[1][j*nPhi+i] = con*0.2;
	    images[2][j*nPhi+i] = con*0.2;
	    of << con << " ";
	}
	of << std::endl;
    }

    write_image(img_string, nPhi, nTheta, images);
} 


__global__ void upsampleParticles
(fReal* particleCoord, fReal* particleVal, float2* weight, size_t numParticles)
{
    // Index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int particleId = index >> 1; // (index / 2)
    int partition = index & 1;	 // (index % 2)

    if (particleId < numParticles) {
	fReal gridLen = M_PI / Rows;
	fReal gPhiId = particleCoord[2 * particleId] * Ratio;
	fReal gThetaId = particleCoord[2 * particleId + 1] * Ratio;

	fReal gTheta = gThetaId * gridLen;
	fReal gPhi = gPhiId * gridLen;

	fReal sinTheta = sinf(gTheta);
	if (sinTheta < 1e-7f)
	    return;

	size_t thetaId = static_cast<size_t>(floorf(gThetaId));

	fReal x1 = cosf(gPhi) * sinTheta; fReal y1 = sinf(gPhi) * sinTheta; fReal z1 = cosf(gTheta);

	fReal theta = (thetaId + 0.5) * gridLen;

	fReal phiRange = 0.5/sinTheta;
	int minPhiId = static_cast<int>(ceilf(gPhiId - phiRange));
	int maxPhiId = static_cast<int>(floorf(gPhiId + phiRange));

	fReal z2 = cosf(theta);
	fReal r = sinf(theta);
	fReal value = particleVal[particleId];

	int begin; int end;
	
	if (partition == 0) {
	    begin = minPhiId; end = static_cast<int>(gPhiId);
	} else {
	    begin = static_cast<int>(gPhiId); end = maxPhiId + 1;
	}
	    
	for (int phiId = begin; phiId < end; phiId++) {
	    fReal phi = phiId * gridLen;
	    fReal x2 = cosf(phi) * r; fReal y2 = sinf(phi) * r;

	    fReal dist2 = powf(fabsf(x1 - x2), 2.f) + powf(fabsf(y1 - y2), 2.f) + powf(fabsf(z1 - z2), 2.f);
	        
	    if (dist2 <= .25f) {
		fReal w = expf(-10*dist2);
		size_t normalizedPhiId = (phiId + Cols) % Cols;
		float2* currentWeight = weight + (thetaId * Cols + normalizedPhiId);
		atomicAdd(&(currentWeight->x), w);
		atomicAdd(&(currentWeight->y), w * value);
	    }
	}
    }
}


__global__ void normalizeThickness
(fReal* thicknessHighRes, fReal* thicknessLowRes, float2* weight, size_t pitch) {
    // Index
    int splitVal = Cols / blockDim.x;
    int threadSequence = blockIdx.x % splitVal;
    int phiId = threadIdx.x + threadSequence * blockDim.x;
    int thetaId = blockIdx.x / splitVal;

    float2* currentWeight = weight + (thetaId * Cols + phiId);
    fReal w = currentWeight->x;
    fReal val = currentWeight->y;

    thicknessHighRes[thetaId * Cols + phiId] = 0.f;
    __syncthreads();

    if (w > 0) {
    	thicknessHighRes[thetaId * Cols + phiId] = val / w;
    } 
    __syncthreads();
    if (w == 0) {
    	// int neighbors[4];
    	// if (thetaId == 0) {
    	//     neighbors[0] = thetaId * Cols + (phiId + Cols / 2) % Cols;
    	// } else {
    	//     neighbors[0] = (thetaId - 1) * Cols + phiId;
    	// }
    	// if (thetaId == Rows - 1) {
    	//     neighbors[1] = thetaId * Cols + (phiId + Cols / 2) % Cols;
    	// } else {
    	//     neighbors[1] = (thetaId + 1) * Cols + phiId;
    	// }
    	// neighbors[2] = thetaId * Cols + (phiId + 1) % Cols;
    	// neighbors[3] = thetaId * Cols + (phiId - 1 + Cols) % Cols;
    	// fReal valn = 0.f;
    	// int nonZero = 0;
    	// for (int i = 0; i < 4; i++) {
    	//     fReal currentNeighbor = thicknessHighRes[neighbors[i]];
    	//     if (currentNeighbor > 0) {
    	// 	valn += currentNeighbor;
    	// 	nonZero++;
    	//     }
    	// }
    	// if (nonZero == 4) {
    	//     thicknessHighRes[thetaId * Cols + phiId] = valn / nonZero;
	// } else {
	    fReal gPhiId = ((fReal)phiId + centeredPhiOffset) / Ratio;
	    fReal gThetaId = ((fReal)thetaId + centeredThetaOffset) / Ratio;
	    thicknessHighRes[thetaId * Cols + phiId] = sampleCentered(thicknessLowRes, gPhiId, gThetaId, pitch);
	// }
    }
}


void KaminoSolver::write_thickness_img(const std::string& s, const int frame)
{
    std::string img_string = std::to_string(frame);
    while (img_string.length() < 4) {
	img_string.insert(0, "0");
    }
    img_string.insert(0, s);
    std::string mat_string = img_string;
    img_string.append(".exr");
    mat_string.append(".txt");

    std::ofstream of(mat_string);

    //    if (frame != 0) {
    if (false) {
	dim3 gridLayout;
	dim3 blockLayout;
	determineLayout(gridLayout, blockLayout, rows, cols);
	resetThickness<<<gridLayout, blockLayout>>>(weightFull);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	
        if (particles->numOfParticles > 0) {
	    determineLayout(gridLayout, blockLayout, 2, particles->numOfParticles);
	    upsampleParticles<<<gridLayout, blockLayout>>>
		(particles->coordGPUThisStep, particles->value, weightFull, particles->numOfParticles);
	    checkCudaErrors(hipGetLastError());
	    checkCudaErrors(hipDeviceSynchronize());
	}
	
	determineLayout(gridLayout, blockLayout, rows, cols);
	normalizeThickness<<<gridLayout, blockLayout>>>
	    (thicknessFull, thickness->getGPUThisStep(), weightFull, thickness->getThisStepPitchInElements());
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	hipMemcpy(thicknessFullCPU, thicknessFull, (cols * rows) * sizeof(fReal), hipMemcpyDeviceToHost);

	std::vector<float> images[3];
	images[0].resize(cols * rows);
	images[1].resize(cols * rows);
	images[2].resize(cols * rows);

	for (size_t j = 0; j < rows; ++j) {
	    for (size_t i = 0; i < cols; ++i) {
		fReal Delta = thicknessFullCPU[j * cols + i];
		if (Delta < 0) { 
		    this->setBroken(true);
		    return;
		} else {
		    images[0][j * cols + i] = Delta * this->H * 5e5;
		    images[1][j * cols + i] = Delta * this->H * 5e5;
		    images[2][j * cols + i] = Delta * this->H * 5e5; // *4
		    of << Delta * this->H * 2<< " ";
		}
	    }
	    of << std::endl;
	}

	write_image(img_string, cols, rows, images);
    } else {
	thickness->copyBackToCPU();

	std::vector<float> images[3];
	images[0].resize(nPhi * nTheta);
	images[1].resize(nPhi * nTheta);
	images[2].resize(nPhi * nTheta);

	for (size_t j = 0; j < nTheta; ++j) {
	    for (size_t i = 0; i < nPhi; ++i) {
		fReal Delta = thickness->getCPUValueAt(i, j);
		if (Delta < 0) {
		    this->setBroken(true);
		}    //		    return;
		//} else {
		images[0][j*nPhi+i] = Delta * this->H * 5e5;
		images[1][j*nPhi+i] = Delta * this->H * 5e5;
		images[2][j*nPhi+i] = Delta * this->H * 5e5; // * 4
		of << Delta * this->H * 2 << " ";
		//}
	    }
	    of << std::endl;
	}

	write_image(img_string, nPhi, nTheta, images);
    }    
}
