# include "../include/KaminoQuantity.cuh"

void KaminoQuantity::copyToGPU() {
    /* 
       Pitch : nPhi * sizeof(float)
       Width : nPhi * sizeof(float)
       Height: nTheta
    */
    checkCudaErrors(hipMemcpy2D(gpuThisStep, thisStepPitch, cpuBuffer, 
				 nPhi * sizeof(float), nPhi * sizeof(float), nTheta, hipMemcpyHostToDevice));
}


void KaminoQuantity::copyBackToCPU() {
    checkCudaErrors(hipMemcpy2D((void*)this->cpuBuffer, nPhi * sizeof(float), (void*)this->gpuThisStep,
				 this->thisStepPitch, nPhi * sizeof(float), nTheta, hipMemcpyDeviceToHost));
}


KaminoQuantity::KaminoQuantity(std::string attributeName, size_t nPhi, size_t nTheta,
			       float phiOffset, float thetaOffset)
    : nPhi(nPhi), nTheta(nTheta), gridLen(M_2PI / nPhi), invGridLen(1.0 / gridLen),
    attrName(attributeName), phiOffset(phiOffset), thetaOffset(thetaOffset) {
    cpuBuffer = new float[nPhi * nTheta];
    checkCudaErrors(hipMallocPitch(&gpuThisStep, &thisStepPitch, nPhi * sizeof(float), nTheta));
    checkCudaErrors(hipMallocPitch(&gpuNextStep, &nextStepPitch, nPhi * sizeof(float), nTheta));
    checkCudaErrors(hipMemset(gpuThisStep, 0, thisStepPitch * nTheta));
    checkCudaErrors(hipMemset(gpuNextStep, 0, nextStepPitch * nTheta));
}


BimocqQuantity::BimocqQuantity(std::string attributeName, size_t nPhi, size_t nTheta,
			       float phiOffset, float thetaOffset)
    : KaminoQuantity(attributeName, nPhi, nTheta, phiOffset, thetaOffset) {
    checkCudaErrors(hipMallocPitch(&gpuInit, &thisStepPitch, nPhi * sizeof(float), nTheta));
    checkCudaErrors(hipMallocPitch(&gpuDelta, &thisStepPitch, nPhi * sizeof(float), nTheta));
    checkCudaErrors(hipMallocPitch(&gpuInitLast, &thisStepPitch, nPhi * sizeof(float), nTheta));
    checkCudaErrors(hipMallocPitch(&gpuDeltaLast, &thisStepPitch, nPhi * sizeof(float), nTheta));
    checkCudaErrors(hipMemset(gpuInit, 0, thisStepPitch * nTheta));
    checkCudaErrors(hipMemset(gpuDelta, 0, thisStepPitch * nTheta));
    checkCudaErrors(hipMemset(gpuInitLast, 0, thisStepPitch * nTheta));
    checkCudaErrors(hipMemset(gpuDeltaLast, 0, thisStepPitch * nTheta));
}


KaminoQuantity::~KaminoQuantity() {
    delete[] cpuBuffer;

    checkCudaErrors(hipFree(gpuThisStep));
    checkCudaErrors(hipFree(gpuNextStep));
}


BimocqQuantity::~BimocqQuantity() {
    checkCudaErrors(hipFree(gpuInit));
    checkCudaErrors(hipFree(gpuDelta));
    checkCudaErrors(hipFree(gpuInitLast));
    checkCudaErrors(hipFree(gpuDeltaLast));
}


std::string KaminoQuantity::getName() {
    return this->attrName;
}


size_t KaminoQuantity::getNPhi() {
    return this->nPhi;
}


size_t KaminoQuantity::getNTheta() {
    return this->nTheta;
}


void KaminoQuantity::swapGPUBuffer() {
    float* tempPtr = this->gpuThisStep;
    this->gpuThisStep = this->gpuNextStep;
    this->gpuNextStep = tempPtr;
}


float KaminoQuantity::getCPUValueAt(size_t phi, size_t theta) {
    return this->accessCPUValueAt(phi, theta);
}


void KaminoQuantity::setCPUValueAt(size_t phi, size_t theta, float val) {
    this->accessCPUValueAt(phi, theta) = val;
}


float& KaminoQuantity::accessCPUValueAt(size_t phi, size_t theta) {
    assert(theta >= 0 && theta < nTheta && phi >= 0 && phi < nPhi);
    return this->cpuBuffer[theta * nPhi + phi];
}


float KaminoQuantity::getThetaOffset() {
    return this->thetaOffset;
}


float KaminoQuantity::getPhiOffset() {
    return this->phiOffset;
}


float* KaminoQuantity::getGPUThisStep() {
    return this->gpuThisStep;
}


float* KaminoQuantity::getGPUNextStep() {
    return this->gpuNextStep;
}


float*& BimocqQuantity::getGPUInit() {
    return this->gpuInit;
}


float*& BimocqQuantity::getGPUDelta() {
    return this->gpuDelta;
}


float*& BimocqQuantity::getGPUInitLast() {
    return this->gpuInitLast;
}


float*& BimocqQuantity::getGPUDeltaLast() {
    return this->gpuDeltaLast;
}


size_t KaminoQuantity::getThisStepPitchInElements() {
    return this->thisStepPitch / sizeof(float);
}


size_t KaminoQuantity::getNextStepPitchInElements() {
    return this->nextStepPitch / sizeof(float);
}