#include "hip/hip_runtime.h"
# include "KaminoSolver.cuh"
# include "../include/KaminoGPU.cuh"
# include "../include/KaminoTimer.cuh"

__constant__ fReal invGridLenGlobal;
static __constant__ size_t nPhiGlobal;
static __constant__ size_t nThetaGlobal;
static __constant__ fReal invRadiusGlobal;
static __constant__ fReal radiusGlobal;
static __constant__ fReal timeStepGlobal;
static __constant__ fReal currentTimeGlobal;
static __constant__ fReal gridLenGlobal;
static __constant__ fReal SGlobal;
static __constant__ fReal MGlobal;
static __constant__ fReal reGlobal;
static __constant__ fReal gGlobal;
static __constant__ fReal DsGlobal;
static __constant__ fReal CrGlobal;
static __constant__ fReal UGlobal;

#define eps 1e-5f

__device__ bool validateCoord(fReal& phi, fReal& theta, size_t& nPhi) {
    bool ret = false;
    // assume theta lies not too far away from the interval [0, nThetaGlobal],
    // otherwise is the step size too large;
    size_t nTheta = nPhi / 2;

    if (theta >= nTheta) {
	theta = nPhi - theta;
	phi += nTheta;
    	ret = !ret;
    }
    if (theta < 0) {
    	theta = -theta;
    	phi += nTheta;
    	ret = !ret;
    }
    if (theta > nTheta || theta < 0)
	printf("Warning: step size too large! theta = %f\n", theta);
    phi = fmod(phi + nPhi, (fReal)nPhi);
    return ret;
}

__device__ fReal kaminoLerp(fReal from, fReal to, fReal alpha)
{
    return (1.0 - alpha) * from + alpha * to;
}

__device__ fReal sampleVPhi(fReal* input, fReal phiRawId, fReal thetaRawId, size_t pitch) {
    fReal phi = phiRawId - vPhiPhiOffset;
    fReal theta = thetaRawId - vPhiThetaOffset;
    // Phi and Theta are now shifted back to origin

    bool isFlippedPole = validateCoord(phi, theta, nPhiGlobal);

    int phiIndex = static_cast<int>(floorf(phi));
    int thetaIndex = static_cast<int>(floorf(theta));
    fReal alphaPhi = phi - static_cast<fReal>(phiIndex);
    fReal alphaTheta = theta - static_cast<fReal>(thetaIndex);
    
    if (thetaIndex == 0 && isFlippedPole) {
	size_t phiLower = (phiIndex) % nPhiGlobal;
	size_t phiHigher = (phiLower + 1) % nPhiGlobal;
	fReal higherBelt = -kaminoLerp(input[phiLower + pitch * thetaIndex],
				       input[phiHigher + pitch * thetaIndex], alphaPhi);

	phiLower = (phiIndex + nPhiGlobal / 2) % nPhiGlobal;
	phiHigher = (phiLower + 1) % nPhiGlobal;

	fReal lowerBelt = kaminoLerp(input[phiLower + pitch * thetaIndex],
				     input[phiHigher + pitch * thetaIndex], alphaPhi);
  
	fReal lerped = kaminoLerp(lowerBelt, higherBelt, alphaTheta);
	return lerped;
    }
    
    if (isFlippedPole) {
	thetaIndex -= 1;
    }
    
    if (thetaIndex == nThetaGlobal - 1) {
	size_t phiLower = (phiIndex) % nPhiGlobal;
	size_t phiHigher = (phiLower + 1) % nPhiGlobal;
	fReal lowerBelt = kaminoLerp(input[phiLower + pitch * thetaIndex],
				     input[phiHigher + pitch * thetaIndex], alphaPhi);
	
	phiLower = (phiIndex + nPhiGlobal / 2) % nPhiGlobal;
	phiHigher = (phiLower + 1) % nPhiGlobal;

	fReal higherBelt = -kaminoLerp(input[phiLower + pitch * thetaIndex],
				       input[phiHigher + pitch * thetaIndex], alphaPhi);

	fReal lerped = kaminoLerp(lowerBelt, higherBelt, alphaTheta);
	if (isFlippedPole)
	    lerped = -lerped;
	return lerped;
    }
  
    size_t phiLower = phiIndex % nPhiGlobal;
    size_t phiHigher = (phiLower + 1) % nPhiGlobal;
    size_t thetaLower = thetaIndex;
    size_t thetaHigher = thetaIndex + 1;

    fReal lowerBelt = kaminoLerp(input[phiLower + pitch * thetaLower],
				 input[phiHigher + pitch * thetaLower], alphaPhi);
    fReal higherBelt = kaminoLerp(input[phiLower + pitch * thetaHigher],
				  input[phiHigher + pitch * thetaHigher], alphaPhi);

    fReal lerped = kaminoLerp(lowerBelt, higherBelt, alphaTheta);
    if (isFlippedPole)
	lerped = -lerped;
    return lerped;
}
    

__device__ fReal sampleVTheta(fReal* input, fReal phiRawId, fReal thetaRawId, size_t pitch) {
    fReal phi = phiRawId - vThetaPhiOffset;
    fReal theta = thetaRawId - vThetaThetaOffset;
    // Phi and Theta are now shifted back to origin

    bool isFlippedPole = validateCoord(phi, theta, nPhiGlobal);

    int phiIndex = static_cast<int>(floorf(phi));
    int thetaIndex = static_cast<int>(floorf(theta));
    fReal alphaPhi = phi - static_cast<fReal>(phiIndex);
    fReal alphaTheta = theta - static_cast<fReal>(thetaIndex);
    
    if (thetaRawId < 0 && thetaRawId > -1 || thetaRawId > nThetaGlobal && thetaRawId < nThetaGlobal + 1 ) {
	thetaIndex -= 1;
	alphaTheta += 1;
    } else if (thetaRawId >= nThetaGlobal + 1 || thetaRawId <= -1) {
    	thetaIndex -= 2;
    }

    if (thetaIndex == 0 && isFlippedPole && thetaRawId > -1) {
    	size_t phiLower = phiIndex % nPhiGlobal;
    	size_t phiHigher = (phiLower + 1) % nPhiGlobal;
    	fReal higherBelt = -kaminoLerp(input[phiLower + pitch * thetaIndex],
    				       input[phiHigher + pitch * thetaIndex], alphaPhi);
	
    	phiLower = (phiLower + nPhiGlobal / 2) % nPhiGlobal;
    	phiHigher = (phiHigher + nPhiGlobal / 2) % nPhiGlobal;
    	fReal lowerBelt = kaminoLerp(input[phiLower + pitch * thetaIndex],
    				     input[phiHigher + pitch * thetaIndex], alphaPhi);

    	alphaTheta = 0.5 * alphaTheta;
    	fReal lerped = kaminoLerp(lowerBelt, higherBelt, alphaTheta);
    	return lerped;
	
    }
    
    if (thetaIndex == nThetaGlobal - 2) {
	size_t phiLower = phiIndex % nPhiGlobal;
	size_t phiHigher = (phiLower + 1) % nPhiGlobal;
	fReal lowerBelt = kaminoLerp(input[phiLower + pitch * thetaIndex],
				     input[phiHigher + pitch * thetaIndex], alphaPhi);

	phiLower = (phiLower + nPhiGlobal / 2) % nPhiGlobal;
	phiHigher = (phiHigher + nPhiGlobal / 2) % nPhiGlobal;
	fReal higherBelt = -kaminoLerp(input[phiLower + pitch * thetaIndex],
				       input[phiHigher + pitch * thetaIndex], alphaPhi);

	alphaTheta = 0.5 * alphaTheta;
	fReal lerped = kaminoLerp(lowerBelt, higherBelt, alphaTheta);
	if (isFlippedPole)
	    lerped = -lerped;
	return lerped;
    }
    
    size_t phiLower = phiIndex % nPhiGlobal;
    size_t phiHigher = (phiLower + 1) % nPhiGlobal;
    size_t thetaLower = thetaIndex;
    size_t thetaHigher = thetaIndex + 1;

    fReal lowerBelt = kaminoLerp(input[phiLower + pitch * thetaLower],
				 input[phiHigher + pitch * thetaLower], alphaPhi);
    fReal higherBelt = kaminoLerp(input[phiLower + pitch * thetaHigher],
				  input[phiHigher + pitch * thetaHigher], alphaPhi);

    fReal lerped = kaminoLerp(lowerBelt, higherBelt, alphaTheta);
    if (isFlippedPole)
	lerped = -lerped;
    return lerped;
}

__device__ fReal sampleCentered(fReal* input, fReal phiRawId, fReal thetaRawId, size_t pitch) {
    fReal phi = phiRawId - centeredPhiOffset;
    fReal theta = thetaRawId - centeredThetaOffset;
    // Phi and Theta are now shifted back to origin

    // if (thetaRawId > nThetaGlobal)
    // 	printf("theta %f\n", theta);
    // TODO change for particles
    bool isFlippedPole = validateCoord(phi, theta, pitch);

    int phiIndex = static_cast<int>(floorf(phi));
    int thetaIndex = static_cast<int>(floorf(theta));
    fReal alphaPhi = phi - static_cast<fReal>(phiIndex);
    fReal alphaTheta = theta - static_cast<fReal>(thetaIndex);

    if (thetaIndex == 0 && isFlippedPole) {
	size_t phiLower = phiIndex % nPhiGlobal;
	size_t phiHigher = (phiLower + 1) % nPhiGlobal;
	fReal higherBelt = kaminoLerp(input[phiLower + pitch * thetaIndex],
				      input[phiHigher + pitch * thetaIndex], alphaPhi);

	phiLower = (phiLower + nPhiGlobal / 2) % nPhiGlobal;
	phiHigher = (phiHigher + nPhiGlobal / 2) % nPhiGlobal;
	fReal lowerBelt = kaminoLerp(input[phiLower + pitch * thetaIndex],
				     input[phiHigher + pitch * thetaIndex], alphaPhi);

	fReal lerped = kaminoLerp(lowerBelt, higherBelt, alphaTheta);
	return lerped;
    }
    
    if (isFlippedPole) {
	thetaIndex -= 1;
    }
    
    if (thetaIndex == nThetaGlobal - 1) {
	size_t phiLower = phiIndex % nPhiGlobal;
	size_t phiHigher = (phiLower + 1) % nPhiGlobal;
	fReal lowerBelt = kaminoLerp(input[phiLower + pitch * thetaIndex],
				     input[phiHigher + pitch * thetaIndex], alphaPhi);

	phiLower = (phiLower + nPhiGlobal / 2) % nPhiGlobal;
	phiHigher = (phiHigher + nPhiGlobal / 2) % nPhiGlobal;
	fReal higherBelt = kaminoLerp(input[phiLower + pitch * thetaIndex],
				      input[phiHigher + pitch * thetaIndex], alphaPhi);

	fReal lerped = kaminoLerp(lowerBelt, higherBelt, alphaTheta);
	return lerped;
    }

    size_t phiLower = phiIndex % nPhiGlobal;
    size_t phiHigher = (phiLower + 1) % nPhiGlobal;
    size_t thetaLower = thetaIndex;
    size_t thetaHigher = thetaIndex + 1;

    fReal lowerBelt = kaminoLerp(input[phiLower + pitch * thetaLower],
				 input[phiHigher + pitch * thetaLower], alphaPhi);
    fReal higherBelt = kaminoLerp(input[phiLower + pitch * thetaHigher],
				  input[phiHigher + pitch * thetaHigher], alphaPhi);

    fReal lerped = kaminoLerp(lowerBelt, higherBelt, alphaTheta);
    return lerped;
}

__global__ void advectionVPhiKernel
(fReal* attributeOutput, fReal* velPhi, fReal* velTheta, size_t pitch)
{
    // Index
    int splitVal = nPhiGlobal / blockDim.x;
    int threadSequence = blockIdx.x % splitVal;
    int phiId = threadIdx.x + threadSequence * blockDim.x;
    int thetaId = blockIdx.x / splitVal;
    
    // Coord in phi-theta space
    fReal gPhiId = (fReal)phiId + vPhiPhiOffset;
    fReal gThetaId = (fReal)thetaId + vPhiThetaOffset;
    fReal gTheta = gThetaId * gridLenGlobal;
	
    // Sample the speed
    fReal guTheta = sampleVTheta(velTheta, gPhiId, gThetaId, pitch);
    fReal guPhi = velPhi[thetaId * pitch + phiId];
       
    fReal cofTheta = timeStepGlobal * invRadiusGlobal * invGridLenGlobal;
    fReal cofPhi = cofTheta / sinf(gTheta);
    
    fReal deltaPhi = guPhi * cofPhi;
    fReal deltaTheta = guTheta * cofTheta;

# ifdef RUNGE_KUTTA
    // Traced halfway in phi-theta space
    fReal midPhiId = gPhiId - 0.5 * deltaPhi;
    fReal midThetaId = gThetaId - 0.5 * deltaTheta;

    fReal muPhi = sampleVPhi(velPhi, midPhiId, midThetaId, pitch);
    fReal muTheta = sampleVTheta(velTheta, midPhiId, midThetaId, pitch);

    // fReal averuPhi = 0.5 * (muPhi + guPhi);
    // fReal averuTheta = 0.5 * (muTheta + guTheta);

    deltaPhi = muPhi * cofPhi;
    deltaTheta = muTheta * cofTheta;
# endif

    fReal pPhiId = gPhiId - deltaPhi;
    fReal pThetaId = gThetaId - deltaTheta;

    fReal advectedVal = sampleVPhi(velPhi, pPhiId, pThetaId, pitch);

    attributeOutput[thetaId * pitch + phiId] = advectedVal;
};

__global__ void advectionVThetaKernel
(fReal* attributeOutput, fReal* velPhi, fReal* velTheta, size_t pitch)
{
    // Index
    int splitVal = nPhiGlobal / blockDim.x;
    int threadSequence = blockIdx.x % splitVal;
    int phiId = threadIdx.x + threadSequence * blockDim.x;
    int thetaId = blockIdx.x / splitVal;
    // Coord in phi-theta space
    fReal gPhiId = (fReal)phiId + vThetaPhiOffset;
    fReal gThetaId = (fReal)thetaId + vThetaThetaOffset;
    fReal gTheta = gThetaId * gridLenGlobal;

    // Sample the speed
    int thetaSouthId = thetaId + 1;
    int phiEastId = (phiId + 1) % nPhiGlobal;
    fReal u0 = velPhi[thetaId * pitch + phiId];
    fReal u1 = velPhi[thetaId * pitch + phiEastId];
    fReal u2 = velPhi[thetaSouthId * pitch + phiId];
    fReal u3 = velPhi[thetaSouthId * pitch + phiEastId];
    fReal guPhi = 0.25 * (u0 + u1 + u2 + u3);
    fReal guTheta = velTheta[thetaId * pitch + phiId];
    
    fReal cofTheta = timeStepGlobal * invRadiusGlobal * invGridLenGlobal;
    fReal cofPhi = cofTheta / sinf(gTheta);
    
    fReal deltaPhi = guPhi * cofPhi;
    fReal deltaTheta = guTheta * cofTheta;

# ifdef RUNGE_KUTTA
    // Traced halfway in phi-theta space
    fReal midPhiId = gPhiId - 0.5 * deltaPhi;
    fReal midThetaId = gThetaId - 0.5 * deltaTheta;
    fReal muPhi = sampleVPhi(velPhi, midPhiId, midThetaId, pitch);
    fReal muTheta = sampleVTheta(velTheta, midPhiId, midThetaId, pitch);

    // fReal averuPhi = 0.5 * (muPhi + guPhi);
    // fReal averuTheta = 0.5 * (muTheta + guTheta);

    deltaPhi = muPhi * cofPhi;
    deltaTheta = muTheta * cofTheta;
# endif

    fReal pPhiId = gPhiId - deltaPhi;
    fReal pThetaId = gThetaId - deltaTheta;

    fReal advectedVal = sampleVTheta(velTheta, pPhiId, pThetaId, pitch);

    attributeOutput[thetaId * pitch + phiId] = advectedVal;
}

__global__ void advectionCentered
(fReal* attributeOutput, fReal* attributeInput, fReal* velPhi, fReal* velTheta, size_t nPitchInElements)
{
    // Index
    int splitVal = nPhiGlobal / blockDim.x;
    int threadSequence = blockIdx.x % splitVal;
    int phiId = threadIdx.x + threadSequence * blockDim.x;
    int thetaId = blockIdx.x / splitVal;
    // Coord in phi-theta space
    fReal gPhiId = (fReal)phiId + centeredPhiOffset;
    fReal gThetaId = (fReal)thetaId + centeredThetaOffset;
    fReal gTheta = gThetaId * gridLenGlobal;
    
    // Sample the speed
    // fReal guPhi = sampleVPhi(velPhi, gPhiId, gThetaId, nPitchInElements);
    fReal guTheta = sampleVTheta(velTheta, gPhiId, gThetaId, nPitchInElements);
    fReal guPhi = 0.5 * (velPhi[thetaId * nPitchInElements + phiId] +
    			 velPhi[thetaId * nPitchInElements + (phiId + 1) % nPhiGlobal]);

    fReal cofTheta = timeStepGlobal * invRadiusGlobal * invGridLenGlobal;
    fReal cofPhi = cofTheta / sinf(gTheta);

    fReal deltaPhi = guPhi * cofPhi;
    fReal deltaTheta = guTheta * cofTheta;

# ifdef RUNGE_KUTTA
    // Traced halfway in phi-theta space
    fReal midPhiId = gPhiId - 0.5 * deltaPhi;
    fReal midThetaId = gThetaId - 0.5 * deltaTheta;
    
    fReal muPhi = sampleVPhi(velPhi, midPhiId, midThetaId, nPitchInElements);
    fReal muTheta = sampleVTheta(velTheta, midPhiId, midThetaId, nPitchInElements);

    deltaPhi = muPhi * cofPhi;
    deltaTheta = muTheta * cofTheta;
# endif

    fReal pPhiId = gPhiId - deltaPhi;
    fReal pThetaId = gThetaId - deltaTheta;

    fReal advectedAttribute = sampleCentered(attributeInput, pPhiId, pThetaId, nPitchInElements);
     
    attributeOutput[thetaId * nPitchInElements + phiId] = advectedAttribute;
};


__global__ void advectionAllCentered
(fReal* thicknessOutput, fReal* surfOutput, fReal* thicknessInput, fReal* surfInput, fReal* velPhi, fReal* velTheta, size_t nPitchInElements)
{
    // Index
    int splitVal = nPhiGlobal / blockDim.x;
    int threadSequence = blockIdx.x % splitVal;
    int phiId = threadIdx.x + threadSequence * blockDim.x;
    int thetaId = blockIdx.x / splitVal;
    // Coord in phi-theta space
    fReal gPhiId = (fReal)phiId + centeredPhiOffset;
    fReal gThetaId = (fReal)thetaId + centeredThetaOffset;
    fReal gTheta = gThetaId * gridLenGlobal;
    
    // Sample the speed
    fReal guTheta = sampleVTheta(velTheta, gPhiId, gThetaId, nPitchInElements);
    fReal guPhi = 0.5 * (velPhi[thetaId * nPitchInElements + phiId] +
    			 velPhi[thetaId * nPitchInElements + (phiId + 1) % nPhiGlobal]);

    fReal cofTheta = timeStepGlobal * invRadiusGlobal * invGridLenGlobal;
    fReal cofPhi = cofTheta / sinf(gTheta);

    fReal deltaPhi = guPhi * cofPhi;
    fReal deltaTheta = guTheta * cofTheta;

# ifdef RUNGE_KUTTA
    // Traced halfway in phi-theta space
    fReal midPhiId = gPhiId - 0.5 * deltaPhi;
    fReal midThetaId = gThetaId - 0.5 * deltaTheta;
    
    fReal muPhi = sampleVPhi(velPhi, midPhiId, midThetaId, nPitchInElements);
    fReal muTheta = sampleVTheta(velTheta, midPhiId, midThetaId, nPitchInElements);

    deltaPhi = muPhi * cofPhi;
    deltaTheta = muTheta * cofTheta;
# endif

    fReal pPhiId = gPhiId - deltaPhi;
    fReal pThetaId = gThetaId - deltaTheta;

    fReal advectedThickness = sampleCentered(thicknessInput, pPhiId, pThetaId, nPitchInElements);
    fReal advectedSurf = sampleCentered(surfInput, pPhiId, pThetaId, nPitchInElements);
    
    thicknessOutput[thetaId * nPitchInElements + phiId] = advectedThickness;
    surfOutput[thetaId * nPitchInElements + phiId] = advectedSurf;
};


__global__ void advectionParticles(fReal* output, fReal* velPhi, fReal* velTheta, fReal* input, size_t pitch, size_t numOfParticles)
{
    int particleId = blockIdx.x * blockDim.x + threadIdx.x;

    if (particleId < numOfParticles) {
	fReal phiId = input[2 * particleId];
	fReal thetaId = input[2 * particleId + 1];
	fReal theta = thetaId * gridLenGlobal;
	fReal sinTheta = max(sinf(theta), eps);

	fReal uPhi = sampleVPhi(velPhi, phiId, thetaId, pitch);
	fReal uTheta = sampleVTheta(velTheta, phiId, thetaId, pitch);
    
	fReal cofTheta = timeStepGlobal * invRadiusGlobal * invGridLenGlobal;
	fReal cofPhi = cofTheta / sinTheta;

	fReal deltaPhi = uPhi * cofPhi;
	fReal deltaTheta = uTheta * cofTheta;

# ifdef RUNGE_KUTTA
	// Traced halfway in phi-theta space
	fReal midPhiId = phiId + 0.5 * deltaPhi;
	fReal midThetaId = thetaId + 0.5 * deltaTheta;

	fReal muPhi = sampleVPhi(velPhi, midPhiId, midThetaId, pitch);
	fReal muTheta = sampleVTheta(velTheta, midPhiId, midThetaId, pitch);

	deltaPhi = muPhi * cofPhi;
	deltaTheta = muTheta * cofTheta;

	theta = midThetaId * gridLenGlobal;
	sinTheta = max(sinf(theta), eps);
# endif

	fReal updatedThetaId = thetaId + deltaTheta;
	fReal updatedPhiId = phiId + deltaPhi;

	validateCoord(updatedPhiId, updatedThetaId, nPhiGlobal);
	
	output[2 * particleId] = updatedPhiId;
	output[2 * particleId + 1] = updatedThetaId;
   }
}


void KaminoSolver::advection(fReal& dt) {
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(timeStepGlobal), &dt, sizeof(fReal)));
    advection();
}

void KaminoSolver::advection()
{
    // kernel call goes here
    // Advect Phi
    dim3 gridLayout;
    dim3 blockLayout;
    determineLayout(gridLayout, blockLayout, velPhi->getNTheta(), velPhi->getNPhi());
    advectionVPhiKernel<<<gridLayout, blockLayout>>>
    	(velPhi->getGPUNextStep(), velPhi->getGPUThisStep(), velTheta->getGPUThisStep(), velPhi->getNextStepPitchInElements());    
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Advect Theta
    determineLayout(gridLayout, blockLayout, velTheta->getNTheta(), velTheta->getNPhi());
    advectionVThetaKernel<<<gridLayout, blockLayout>>>
    	(velTheta->getGPUNextStep(), velPhi->getGPUThisStep(), velTheta->getGPUThisStep(), velTheta->getNextStepPitchInElements());
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
 
    // Advect thickness particles
    if (particles->numOfParticles > 0) {
	determineLayout(gridLayout, blockLayout, 1, particles->numOfParticles);
	advectionParticles<<<gridLayout, blockLayout>>>
	    (particles->coordGPUNextStep, velPhi->getGPUThisStep(), velTheta->getGPUThisStep(),
	     particles->coordGPUThisStep, velPhi->getThisStepPitchInElements(), particles->numOfParticles);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
    }
  
    // Advect concentration
    determineLayout(gridLayout, blockLayout, surfConcentration->getNTheta(), surfConcentration->getNPhi());
    advectionCentered<<<gridLayout, blockLayout>>>
	(surfConcentration->getGPUNextStep(), surfConcentration->getGPUThisStep(),
	 velPhi->getGPUThisStep(), velTheta->getGPUThisStep(), surfConcentration->getNextStepPitchInElements());
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // thickness->swapGPUBuffer();
    particles->swapGPUBuffers(); 
    surfConcentration->swapGPUBuffer();
    swapVelocityBuffers();
}

__device__ fReal _root3(fReal x)
{
    fReal s = 1.;
    while (x < 1.)
	{
	    x *= 8.;
	    s *= 0.5;
	}
    while (x > 8.)
	{
	    x *= 0.125;
	    s *= 2.;
	}
    fReal r = 1.5;
    r -= 1. / 3. * (r - x / (r * r));
    r -= 1. / 3. * (r - x / (r * r));
    r -= 1. / 3. * (r - x / (r * r));
    r -= 1. / 3. * (r - x / (r * r));
    r -= 1. / 3. * (r - x / (r * r));
    r -= 1. / 3. * (r - x / (r * r));
    return r * s;
}

__device__ fReal root3(double x)
{
    if (x > 0)
	return _root3(x);
    else if (x < 0)
	return -_root3(-x);
    else
	return 0.0;
}

__device__ fReal solveCubic(fReal a, fReal b, fReal c)
{
    fReal a2 = a * a;
    fReal q = (a2 - 3 * b) / 9.0;
    //q = q >= 0.0 ? q : -q;
    fReal r = (a * (2.0 * a2 - 9.0 * b) + 27.0 * c) / 54.0;

    fReal r2 = r * r;
    fReal q3 = q * q * q;
    fReal A, B;
    if (r2 <= (q3 + eps))
	{
	    double t = r / sqrtf(q3);
	    if (t < -1)
		t = -1;
	    if (t > 1)
		t = 1;
	    t = acosf(t);
	    a /= 3.0;
	    q = -2.0 * sqrtf(q);
	    return q * cosf(t / 3.0) - a;
	}
    else
	{
	    A = -root3(fabsf(r) + sqrtf(r2 - q3));
	    if (r < 0)
		A = -A;

	    B = A == 0 ? 0 : B = q / A;

	    a /= 3.0;
	    return (A + B) - a;
	}
}

//nTheta by nPhi
__global__ void geometricFillKernel
(fReal* intermediateOutputPhi, fReal* intermediateOutputTheta, fReal* velPhiInput, fReal* velThetaInput,
 size_t nPitchInElements)
{
    // Index
    int splitVal = nPhiGlobal / blockDim.x;
    int threadSequence = blockIdx.x % splitVal;
    int gridPhiId = threadIdx.x + threadSequence * blockDim.x;
    int gridThetaId = blockIdx.x / splitVal;
    // Coord in phi-theta space
    fReal gPhiId = (fReal)gridPhiId + centeredPhiOffset;
    fReal gThetaId = (fReal)gridThetaId + centeredThetaOffset;
    fReal gTheta = gThetaId * gridLenGlobal;
    // The factor

    size_t phiLeft = gridPhiId;
    size_t phiRight = (gridPhiId + 1) % nPhiGlobal;
    // u = velPhi
    fReal uPrev = 0.5 * (velPhiInput[phiLeft + nPitchInElements * gridThetaId]
			 + velPhiInput[phiRight + nPitchInElements * gridThetaId]);

    // v = velTheta	
    fReal vPrev;
    if (gridThetaId == 0)
	{
	    size_t oppositePhiIdx = (gridPhiId + nPhiGlobal / 2) % nPhiGlobal;
	    vPrev = 0.75 * velThetaInput[gridPhiId]
		- 0.25 * velThetaInput[oppositePhiIdx];
	}
    else if (gridThetaId == nThetaGlobal - 1)
	{
	    size_t oppositePhiIdx = (gridPhiId + nPhiGlobal / 2) % nPhiGlobal;
	    vPrev = 0.75 * velThetaInput[gridPhiId + nPitchInElements * (gridThetaId - 1)]
		- 0.25 * velThetaInput[oppositePhiIdx + nPitchInElements * (gridThetaId - 1)];
	}
    else
	{
	    vPrev = 0.5 * (velThetaInput[gridPhiId + nPitchInElements * (gridThetaId - 1)]
			   + velThetaInput[gridPhiId + nPitchInElements * gridThetaId]);
	}

	

    fReal G = timeStepGlobal * cosf(gTheta) * invRadiusGlobal / sinf(gTheta);
    fReal uNext;
    if (abs(G) > eps)
	{
	    fReal cof = G * G;
	    fReal A = 0.0;
	    fReal B = (G * vPrev + 1.0) / cof;
	    fReal C = -uPrev / cof;

	    uNext = solveCubic(A, B, C); 
	}
    else
	{
	    uNext = uPrev;
	}

    fReal vNext = vPrev + G * uNext * uNext;

    intermediateOutputPhi[gridPhiId + nPitchInElements * gridThetaId] = uNext;
    intermediateOutputTheta[gridPhiId + nPitchInElements * gridThetaId] = vNext;
}

//nTheta by nPhi
__global__ void assignPhiKernel(fReal* velPhiOutput, fReal* intermediateInputPhi,
				size_t nPitchInElements)
{
    // Index
    int splitVal = nPhiGlobal / blockDim.x;
    int threadSequence = blockIdx.x % splitVal;
    int phiId = threadIdx.x + threadSequence * blockDim.x;
    int thetaId = blockIdx.x / splitVal;

    size_t phigridLeft;
    if (phiId == 0)
	phigridLeft = nPhiGlobal - 1;
    else
	phigridLeft = phiId - 1;
    velPhiOutput[phiId + nPitchInElements * thetaId] =
	0.5 * (intermediateInputPhi[phigridLeft + nPitchInElements * thetaId]
	       + intermediateInputPhi[phiId + nPitchInElements * thetaId]);
}

//nTheta - 1 by nPhi
__global__ void assignThetaKernel(fReal* velThetaOutput, fReal*intermediateInputTheta,
				  size_t nPitchInElements)
{
    // Index
    int splitVal = nPhiGlobal / blockDim.x;
    int threadSequence = blockIdx.x % splitVal;
    int phiId = threadIdx.x + threadSequence * blockDim.x;
    int thetaId = blockIdx.x / splitVal;

    velThetaOutput[phiId + nPitchInElements * thetaId] =
	0.5 * (intermediateInputTheta[phiId + nPitchInElements * thetaId]
	       + intermediateInputTheta[phiId + nPitchInElements * (thetaId + 1)]);
}

void KaminoSolver::geometric()
{
    dim3 gridLayout;
    dim3 blockLayout;
    //intermediate: pressure.this as phi, next as theta

    determineLayout(gridLayout, blockLayout, nTheta, nPhi);
    geometricFillKernel<<<gridLayout, blockLayout>>> 
	(pressure->getGPUThisStep(), pressure->getGPUNextStep(), velPhi->getGPUThisStep(), velTheta->getGPUThisStep(),
	 pressure->getNextStepPitchInElements());
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());



    determineLayout(gridLayout, blockLayout, velPhi->getNTheta(), velPhi->getNPhi());
    assignPhiKernel<<<gridLayout, blockLayout>>>
	(velPhi->getGPUNextStep(), pressure->getGPUThisStep(), velPhi->getNextStepPitchInElements());
    checkCudaErrors(hipGetLastError());


    determineLayout(gridLayout, blockLayout, velTheta->getNTheta(), velTheta->getNPhi());
    assignThetaKernel<<<gridLayout, blockLayout>>>
	(velTheta->getGPUNextStep(), pressure->getGPUNextStep(), velTheta->getNextStepPitchInElements());
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());



    swapVelocityBuffers();
}

__global__ void crKernel(fReal *d_a, fReal *d_b, fReal *d_c, fReal *d_d, fReal *d_x);

// div(u) at cell center
__global__ void comDivergenceKernel
(fReal* div, fReal* velPhi, fReal* velTheta,
 size_t velPhiPitchInElements, size_t velThetaPitchInElements)
{
    int splitVal = nPhiGlobal / blockDim.x;
    int threadSequence = blockIdx.x % splitVal;
    int phiId = threadIdx.x + threadSequence * blockDim.x;
    int thetaId = blockIdx.x / splitVal;

    fReal thetaCoord = ((fReal)thetaId + centeredThetaOffset) * gridLenGlobal;

    fReal uEast = 0.0;
    fReal uWest = 0.0;
    fReal vNorth = 0.0;
    fReal vSouth = 0.0;

    fReal halfStep = 0.5 * gridLenGlobal;

    fReal thetaSouth = thetaCoord + halfStep;
    fReal thetaNorth = thetaCoord - halfStep;

    int phiIdWest = phiId;
    int phiIdEast = (phiIdWest + 1) % nPhiGlobal;

    uWest = velPhi[thetaId * velPhiPitchInElements + phiIdWest];
    uEast = velPhi[thetaId * velPhiPitchInElements + phiIdEast];
    
    if (thetaId != 0) {
	size_t thetaNorthIdx = thetaId - 1;
	vNorth = velTheta[thetaNorthIdx * velThetaPitchInElements + phiId];
    } 
    if (thetaId != nThetaGlobal - 1) {
	size_t thetaSouthIdx = thetaId;
	vSouth = velTheta[thetaSouthIdx * velThetaPitchInElements + phiId];
    }
    // otherwise sin(theta) = 0;

    fReal invGridSine = 1.0 / sinf(thetaCoord);
    fReal sinNorth = sinf(thetaNorth);
    fReal sinSouth = sinf(thetaSouth);
    fReal factor = invGridSine * invRadiusGlobal * invGridLenGlobal;
    fReal termTheta = factor * (vSouth * sinSouth - vNorth * sinNorth);
    fReal termPhi = factor * (uEast - uWest);

    fReal f = termTheta + termPhi;

    div[thetaId * nPhiGlobal + phiId] = f;
}

__global__ void applyforcevelthetaNoViscosityKernel
(fReal* velThetaOutput, fReal* velThetaInput, fReal* velPhi, fReal* thickness,
 fReal* concentration,  size_t pitch) {
    int splitVal = nPhiGlobal / blockDim.x;
    int threadSequence = blockIdx.x % splitVal;
    int phiId = threadIdx.x + threadSequence * blockDim.x;
    int thetaId = blockIdx.x / splitVal;

    // Coord in phi-theta space
    fReal gTheta = ((fReal)thetaId + vThetaThetaOffset) * gridLenGlobal;

    int thetaSouthId = thetaId + 1;
    int phiEastId = (phiId + 1) % nPhiGlobal;
    
    // -   +  v0 +  -
    // d0  u0   u1  d1
    // v3  +  v1 +  v4
    // d2  u2   u3  d3
    // -   +  v2 +  -
    //
    // v1 is the current velTheta
    fReal u0 = velPhi[thetaId * pitch + phiId];
    fReal u1 = velPhi[thetaId * pitch + phiEastId];
    fReal u2 = velPhi[thetaSouthId * pitch + phiId];
    fReal u3 = velPhi[thetaSouthId * pitch + phiEastId];
    fReal v1 = velThetaInput[thetaId * pitch + phiId];

    // values at centered grid
    fReal GammaNorth = concentration[thetaId * pitch + phiId];
    fReal GammaSouth = concentration[thetaSouthId * pitch + phiId];
    fReal DeltaNorth = thickness[thetaId * pitch + phiId];
    fReal DeltaSouth = thickness[thetaSouthId * pitch + phiId];

    // values at vTheta grid
    fReal Delta = 0.5 * (DeltaNorth + DeltaSouth);
    fReal invDelta = 1. / Delta;
    fReal uPhi = 0.25 * (u0 + u1 + u2 + u3);
    
    // trigonometric function
    fReal sinTheta = sinf(gTheta);
    fReal cscTheta = 1. / sinTheta;
    fReal cosTheta = cosf(gTheta);
    fReal cotTheta = cosTheta * cscTheta;

    // pGpy = \frac{\partial\Gamma}{\partial\theta};
    fReal pGpy = invGridLenGlobal * (GammaSouth - GammaNorth);

    // force terms
    fReal f1 = uPhi * uPhi * cotTheta * invRadiusGlobal;

    fReal f3 = -2 * MGlobal * invDelta * invRadiusGlobal * pGpy;

    fReal vAir = 0.0;
    fReal f6 = vAir - v1;
    // fReal f7 = gGlobal * sinTheta;
    fReal f7 = 0.0;

    // output
    velThetaOutput[thetaId * pitch + phiId] = (v1 + timeStepGlobal * (f1 + f3 + CrGlobal * vAir + f7))
	/ (1.0 + CrGlobal * timeStepGlobal);
}

__global__ void applyforcevelthetaKernel
(fReal* velThetaOutput, fReal* velThetaInput, fReal* velPhi, fReal* thickness,
 fReal* concentration, fReal* divCentered, size_t pitch) {
    int splitVal = nPhiGlobal / blockDim.x;
    int threadSequence = blockIdx.x % splitVal;
    int phiId = threadIdx.x + threadSequence * blockDim.x;
    int thetaId = blockIdx.x / splitVal;

    // Coord in phi-theta space
    fReal gPhi = ((fReal)phiId + vPhiPhiOffset) * gridLenGlobal;
    fReal gTheta = ((fReal)thetaId + vThetaThetaOffset) * gridLenGlobal;

    int thetaNorthId = thetaId;
    int thetaSouthId = thetaId + 1;
    int phiWestId = (phiId - 1 + nPhiGlobal) % nPhiGlobal;
    int phiEastId = (phiId + 1) % nPhiGlobal;
    
    // -   +  v0 +  -
    // d0  u0   u1  d1
    // v3  +  v1 +  v4
    // d2  u2   u3  d3
    // -   +  v2 +  -
    //
    // v1 is the current velTheta
    fReal u0 = velPhi[thetaId * pitch + phiId];
    fReal u1 = velPhi[thetaId * pitch + phiEastId];
    fReal u2 = velPhi[thetaSouthId * pitch + phiId];
    fReal u3 = velPhi[thetaSouthId * pitch + phiEastId];

    // values at centered grid
    fReal divNorth = divCentered[thetaId * nPhiGlobal + phiId];
    fReal divSouth = divCentered[thetaSouthId * nPhiGlobal + phiId];
    fReal GammaNorth = concentration[thetaId * pitch + phiId];
    fReal GammaSouth = concentration[thetaSouthId * pitch + phiId];
    fReal DeltaNorth = thickness[thetaId * pitch + phiId];
    fReal DeltaSouth = thickness[thetaSouthId * pitch + phiId];

    // values at vTheta grid
    fReal div = 0.5 * (divNorth + divSouth);
    fReal Delta = 0.5 * (DeltaNorth + DeltaSouth);
    fReal invDelta = 1. / Delta;
    fReal uPhi = 0.25 * (u0 + u1 + u2 + u3);

    // pDpx = \frac{\partial\Delta}{\partial\phi}
    fReal d0 = thickness[thetaId * pitch + phiWestId];
    fReal d1 = thickness[thetaId * pitch + phiEastId];
    fReal d2 = thickness[thetaSouthId * pitch + phiWestId];
    fReal d3 = thickness[thetaSouthId * pitch + phiEastId];
    fReal pDpx = 0.25 * invGridLenGlobal * (d1 + d3 - d0 - d2);
    fReal pDpy = invGridLenGlobal * (DeltaSouth - DeltaNorth);

    // pvpy = \frac{\partial u_theta}{\partial\theta}
    fReal v0 = 0.0;
    fReal v1 = velThetaInput[thetaId * pitch + phiId];
    fReal v2 = 0.0;
    fReal v3 = velThetaInput[thetaId * pitch + phiWestId];
    fReal v4 = velThetaInput[thetaId * pitch + phiEastId];
    if (thetaId != 0) {
	size_t thetaNorthId = thetaId - 1;
	v0 = velThetaInput[thetaNorthId * pitch + phiId];
    } else {
	size_t oppositePhiId = (phiId + nPhiGlobal / 2) % nPhiGlobal;
	v0 = 0.5 * (velThetaInput[thetaId * pitch + phiId] -
		    velThetaInput[thetaId * pitch + oppositePhiId]);
    }
    if (thetaId != nThetaGlobal - 2) {
	v2 = velThetaInput[thetaSouthId * pitch + phiId];
    } else {
	size_t oppositePhiId = (phiId + nPhiGlobal / 2) % nPhiGlobal;
	v2 = 0.5 * (velThetaInput[thetaId * pitch + phiId] -
		    velThetaInput[thetaId * pitch + oppositePhiId]);
    }

    fReal pvpy = 0.5 * invGridLenGlobal * (v2 - v0);
    fReal pvpyNorth = invGridLenGlobal * (v1 - v0);
    fReal pvpySouth = invGridLenGlobal * (v2 - v1);
	
    // pvpx = \frac{\partial u_theta}{\partial\phi}    
    fReal pvpx = 0.5 * invGridLenGlobal * (v4 - v3);
    
    // pupy = \frac{\partial u_phi}{\partial\theta}
    fReal pupy = 0.5 * invGridLenGlobal * (u2 + u3 - u0 - u1);

    // pupx = \frac{\partial u_phi}{\partial\phi}
    fReal pupx = 0.5 * invGridLenGlobal * (u1 + u3 - u0 - u2);

    // pupxy = \frac{\partial^2u_\phi}{\partial\theta\partial\phi}
    fReal pupxy = invGridLenGlobal * invGridLenGlobal * (u0 + u3 - u1 - u2);

    // pvpxx = \frac{\partial^2u_\theta}{\partial\phi^2}    
    fReal pvpxx = invGridLenGlobal * invGridLenGlobal * (v3 + v4 - 2 * v1);
    
    // trigonometric function
    fReal sinTheta = sinf(gTheta);
    fReal cscTheta = 1. / sinTheta;
    fReal cosTheta = cosf(gTheta);
    fReal cotTheta = cosTheta * cscTheta;

    // stress
    // TODO: laplace term
    fReal sigma11North = 0. +  2 * (invRadiusGlobal * pvpyNorth + divNorth);
    fReal sigma11South = 0. +  2 * (invRadiusGlobal * pvpySouth + divSouth);
    
    // fReal sigma11 = 0. +  2 * invRadiusGlobal * pvpy + 2 * div;
    fReal sigma22 = -2 * invRadiusGlobal * pvpy + 4 * div;
    fReal sigma12 = invRadiusGlobal * (cscTheta * pvpx + pupy - uPhi * cotTheta);

    // psspy = \frac{\partial}{\partial\theta}(\sin\theta\sigma_{11})
    fReal halfStep = 0.5 * gridLenGlobal;    
    fReal thetaSouth = gTheta + halfStep;
    fReal thetaNorth = gTheta - halfStep;    
    fReal sinNorth = sinf(thetaNorth);
    fReal sinSouth = sinf(thetaSouth);    
    fReal psspy = invGridLenGlobal * (sigma11South * sinSouth - sigma11North * sinNorth);
    
    // pspx = \frac{\partial\sigma_{12}}{\partial\phi}
    fReal pspx = invRadiusGlobal * (cscTheta * pvpxx + pupxy - cotTheta * pupx);

    // pGpy = \frac{\partial\Gamma}{\partial\theta};
    fReal pGpy = invGridLenGlobal * (GammaSouth - GammaNorth);

    // force terms
    fReal f1 = uPhi * uPhi * cotTheta * invRadiusGlobal;
    fReal f2 = reGlobal * invRadiusGlobal * cscTheta * invDelta * pDpx * sigma12;
    fReal f3 = -2 * MGlobal * invDelta * invRadiusGlobal * pGpy;
    fReal f4 = reGlobal * invDelta * invRadiusGlobal * pDpy * 2 * (div + invRadiusGlobal * pvpy);
    fReal f5 = reGlobal * invRadiusGlobal * cscTheta * (psspy + pspx - cosTheta * sigma22);

    // fReal f7 = gGlobal * sinTheta;
    fReal f7 = 0.0; 		// Van der Waals
    fReal vAir = 0.0;
    fReal f6 = CrGlobal * (vAir - v1);
    
    // output
    fReal result = (v1 + timeStepGlobal * (f1 + f2 + f3 + f4 + f5 + CrGlobal * vAir + f7))
	/ (1.0 + CrGlobal * timeStepGlobal);
    if (fabsf(result) < eps)
	result = 0.f;
    velThetaOutput[thetaId * pitch + phiId] = result;
}


__global__ void applyforcevelphiNoViscosityKernel
(fReal* velPhiOutput, fReal* velTheta, fReal* velPhiInput, fReal* thickness,
 fReal* concentration, size_t pitch) {
    int splitVal = nPhiGlobal / blockDim.x;
    int threadSequence = blockIdx.x % splitVal;
    int phiId = threadIdx.x + threadSequence * blockDim.x;
    int thetaId = blockIdx.x / splitVal;

    // Coord in phi-theta space
    fReal gPhi = ((fReal)phiId + vPhiPhiOffset) * gridLenGlobal;
    fReal gTheta = ((fReal)thetaId + vPhiThetaOffset) * gridLenGlobal;

    int phiWestId = (phiId - 1 + nPhiGlobal) % nPhiGlobal;
    int thetaNorthId = thetaId - 1;

    // values at centered grid
    fReal DeltaWest = thickness[thetaId * pitch + phiWestId];
    fReal DeltaEast = thickness[thetaId * pitch + phiId];
    fReal GammaWest = concentration[thetaId * pitch + phiWestId];
    fReal GammaEast = concentration[thetaId * pitch + phiId];
    
    // |  d0 u3 d2 |
    // +  v0 +  v1 +
    // u0    u1    u2
    // +  v2 +  v3 + 
    // |  d1 u4 d3 |
    //
    // u1 is the current velPhi
    fReal v0 = 0.0;
    fReal v1 = 0.0;
    fReal v2 = 0.0;
    fReal v3 = 0.0;
    fReal u1 = velPhiInput[thetaId * pitch + phiId];
    
    if (thetaId != 0) {
	v0 = velTheta[thetaNorthId * pitch + phiWestId];
	v1 = velTheta[thetaNorthId * pitch + phiId];
    } else {
	v0 = 0.5 * (velTheta[thetaId * pitch + phiWestId] -
		    velTheta[thetaId * pitch + (phiWestId + nPhiGlobal / 2) % nPhiGlobal]);
	v1 = 0.5 * (velTheta[thetaId * pitch + phiId] -
		    velTheta[thetaId * pitch + (phiId + nPhiGlobal / 2) % nPhiGlobal]);
    }
    if (thetaId != nThetaGlobal - 1) {
	v2 = velTheta[thetaId * pitch + phiWestId];
	v3 = velTheta[thetaId * pitch + phiId];
    } else {
	v2 = 0.5 * (velTheta[thetaNorthId * pitch + phiWestId] -
		    velTheta[thetaNorthId * pitch + (phiWestId + nPhiGlobal / 2) % nPhiGlobal]);
	v3 = 0.5 * (velTheta[thetaNorthId * pitch + phiId] -
		    velTheta[thetaNorthId * pitch + (phiId + nPhiGlobal / 2) % nPhiGlobal]);
    }
    
    // values at uPhi grid
    fReal vTheta = 0.25 * (v0 + v1 + v2 + v3);    
    fReal Delta = 0.5 * (DeltaWest + DeltaEast);
    fReal invDelta = 1. / Delta;
    
    // pGpx = \frac{\partial\Gamma}{\partial\phi};
    fReal pGpx = invGridLenGlobal * (GammaEast - GammaWest);

    // trigonometric function
    fReal sinTheta = sinf(gTheta);
    fReal cscTheta = 1. / sinTheta;
    fReal cosTheta = cosf(gTheta);
    fReal cotTheta = cosTheta * cscTheta;
    
    // force terms
    fReal f1 = -vTheta * u1 * cotTheta * invRadiusGlobal;
    fReal f3 = -2 * MGlobal * invDelta * invRadiusGlobal * cscTheta * pGpx;

    fReal uAir = 0.0;
    if (currentTimeGlobal < 5)
	uAir = 20.f * (M_hPI - gTheta) * expf(-10 * powf(fabsf(gTheta - M_hPI), 2.f)) * radiusGlobal * cosf(gPhi) / UGlobal;
    fReal f6 = uAir - u1;

    // output
    velPhiOutput[thetaId * pitch + phiId] = (u1 + timeStepGlobal * (f3 + CrGlobal * uAir))
	/ (1.0 + (CrGlobal + vTheta * invRadiusGlobal * cotTheta) * timeStepGlobal);
}


__global__ void applyforcevelphiKernel
(fReal* velPhiOutput, fReal* velTheta, fReal* velPhiInput, fReal* thickness,
 fReal* concentration, fReal* divCentered, size_t pitch) {
    int splitVal = nPhiGlobal / blockDim.x;
    int threadSequence = blockIdx.x % splitVal;
    int phiId = threadIdx.x + threadSequence * blockDim.x;
    int thetaId = blockIdx.x / splitVal;

    // Coord in phi-theta space
    fReal gPhi = ((fReal)phiId + vPhiPhiOffset) * gridLenGlobal;
    fReal gTheta = ((fReal)thetaId + vPhiThetaOffset) * gridLenGlobal;

    int phiWestId = (phiId - 1 + nPhiGlobal) % nPhiGlobal;
    int thetaNorthId = thetaId - 1;
    int thetaSouthId = thetaId + 1;

    // values at centered grid
    fReal divWest = divCentered[thetaId * nPhiGlobal + phiWestId];
    fReal divEast = divCentered[thetaId * nPhiGlobal + phiId];
    fReal DeltaWest = thickness[thetaId * pitch + phiWestId];
    fReal DeltaEast = thickness[thetaId * pitch + phiId];
    fReal GammaWest = concentration[thetaId * pitch + phiWestId];
    fReal GammaEast = concentration[thetaId * pitch + phiId];
    
    // |  d0 u3 d2 |
    // +  v0 +  v1 +
    // u0    u1    u2
    // +  v2 +  v3 + 
    // |  d1 u4 d3 |
    //
    // u1 is the current velPhi
    fReal v0 = 0.0;
    fReal v1 = 0.0;
    fReal v2 = 0.0;
    fReal v3 = 0.0;
    if (thetaId != 0) {
	v0 = velTheta[thetaNorthId * pitch + phiWestId];
	v1 = velTheta[thetaNorthId * pitch + phiId];
    } else {
	v0 = 0.5 * (velTheta[thetaId * pitch + phiWestId] -
		    velTheta[thetaId * pitch + (phiWestId + nPhiGlobal / 2) % nPhiGlobal]);
	v1 = 0.5 * (velTheta[thetaId * pitch + phiId] -
		    velTheta[thetaId * pitch + (phiId + nPhiGlobal / 2) % nPhiGlobal]);
    }
    if (thetaId != nThetaGlobal - 1) {
	v2 = velTheta[thetaId * pitch + phiWestId];
	v3 = velTheta[thetaId * pitch + phiId];
    } else {
	v2 = 0.5 * (velTheta[thetaNorthId * pitch + phiWestId] -
		    velTheta[thetaNorthId * pitch + (phiWestId + nPhiGlobal / 2) % nPhiGlobal]);
	v3 = 0.5 * (velTheta[thetaNorthId * pitch + phiId] -
		    velTheta[thetaNorthId * pitch + (phiId + nPhiGlobal / 2) % nPhiGlobal]);
    }
    
    // values at uPhi grid
    fReal Delta = 0.5 * (DeltaWest + DeltaEast);
    fReal invDelta = 1. / Delta;
    fReal div = 0.5 * (divWest + divEast);
    fReal vTheta = 0.25 * (v0 + v1 + v2 + v3);

    // pvpx = \frac{\partial u_theta}{\partial\phi}
    fReal pvpx = 0.5 * invGridLenGlobal * (v1 + v3 - v0 - v2);

    // pvpy = \frac{\partial u_theta}{\partial\theta}
    fReal pvpyWest = invGridLenGlobal * (v2 - v0);
    fReal pvpyEast = invGridLenGlobal * (v3 - v1);
    fReal pvpy = 0.5 * invGridLenGlobal * (v2 + v3 - v0 - v1);

    // pupy = \frac{\partial u_phi}{\partial\theta}
    fReal pupyNorth = 0.0;
    fReal pupySouth = 0.0;
    fReal u1 = velPhiInput[thetaId * pitch + phiId];
    fReal u3 = 0.0;
    fReal u4 = 0.0;
    // actually pupyNorth != 0 at theta == 0, but pupyNorth appears only
    // in sinNorth * pupyNorth, and sinNorth = 0 at theta == 0    
    if (thetaId != 0) {
        u3 = velPhiInput[thetaNorthId * pitch + phiId];
	pupyNorth = invGridLenGlobal * (u1 - u3);
    } else {
	size_t oppositePhiId = (phiId + nPhiGlobal / 2) % nPhiGlobal;
	u3 = -velPhiInput[thetaId * pitch + oppositePhiId];
    }
    // actually pupySouth != 0 at theta == \pi, but pupySouth appears only
    // in sinSouth * pupySouth, and sinSouth = 0 at theta == \pi   
    if (thetaId != nThetaGlobal - 1) {
	u4 = velPhiInput[thetaSouthId * pitch + phiId];
	pupySouth = invGridLenGlobal * (u4 - u1);
    } else {
	size_t oppositePhiId = (phiId + nPhiGlobal / 2) % nPhiGlobal;
	u4 = -velPhiInput[thetaId * pitch + oppositePhiId];
    }
    fReal pupy = 0.5 * invGridLenGlobal * (u4 - u3);

    // pGpx = \frac{\partial\Gamma}{\partial\phi};
    fReal pGpx = invGridLenGlobal * (GammaEast - GammaWest);

    // trigonometric function
    fReal sinTheta = sinf(gTheta);
    fReal cscTheta = 1. / sinTheta;
    fReal cosTheta = cosf(gTheta);
    fReal cotTheta = cosTheta * cscTheta;
    
    // stress
    // TODO: laplace term
    fReal sigma12 = invRadiusGlobal * (cscTheta * pvpx + pupy - u1 * cotTheta);

    // pDpx = \frac{\partial\Delta}{\partial\phi}
    fReal pDpx = invGridLenGlobal * (DeltaEast - DeltaWest);

    // pDpy = \frac{\partial\Delta}{\partial\theta}
    // TODO: do we need to average the thickness value at the pole?
    fReal pDpy = 0.0;
    fReal d0 = 0.0;
    fReal d1 = 0.0;
    fReal d2 = 0.0;
    fReal d3 = 0.0;
    if (thetaId != 0) {
	d0 = thickness[thetaNorthId * pitch + phiWestId];
	d2 = thickness[thetaNorthId * pitch + phiId];
    } else {
	d0 = thickness[thetaId * pitch + (phiWestId + nPhiGlobal / 2) % nPhiGlobal];
	d2 = thickness[thetaId * pitch + (phiId + nPhiGlobal / 2) % nPhiGlobal];
    }
    if (thetaId != nThetaGlobal - 1) {
	d1 = thickness[thetaSouthId * pitch + phiWestId];
	d3 = thickness[thetaSouthId * pitch + phiId];
    } else {
	d1 = thickness[thetaId * pitch + (phiWestId + nPhiGlobal / 2) % nPhiGlobal];
	d3 = thickness[thetaId * pitch + (phiId + nPhiGlobal / 2) % nPhiGlobal];
    }
    pDpy = 0.25 * invGridLenGlobal * (d1 + d3 - d0 - d2);
    
    // psspy = \frac{\partial}{\partial\theta}(\sin\theta\sigma_{12})
    fReal halfStep = 0.5 * gridLenGlobal;    
    fReal thetaSouth = gTheta + halfStep;
    fReal thetaNorth = gTheta - halfStep;    
    fReal sinNorth = sinf(thetaNorth);
    fReal sinSouth = sinf(thetaSouth);
    fReal cosNorth = cosf(thetaNorth);
    fReal cosSouth = cosf(thetaSouth);
    // TODO: uncertain about the definintion of u_\phi at both poles
    fReal uNorth = 0.5 * (u3 + u1);
    fReal uSouth = 0.5 * (u1 + u4);
    fReal psspy = invRadiusGlobal * invGridLenGlobal * (invGridLenGlobal * (v0 + v3 - v1 - v2) +
							sinSouth * pupySouth - sinNorth * pupyNorth -
							cosSouth * uSouth + cosNorth * uNorth);
    
    // pspx = \frac{\partial\sigma_{22}}{\partial\phi}
    fReal sigma22West = 2 * (2 * divWest - invRadiusGlobal * pvpyWest);
    fReal sigma22East = 2 * (2 * divEast - invRadiusGlobal * pvpyEast);
    fReal pspx = invGridLenGlobal * (sigma22East - sigma22West);
    
    // force terms
    // fReal f1 = -vTheta * u1 * cotTheta * invRadiusGlobal;
    fReal f2 = reGlobal * invRadiusGlobal * invDelta * pDpy * sigma12;
    fReal f3 = -2 * MGlobal * invDelta * invRadiusGlobal * cscTheta * pGpx;
    fReal f4 = reGlobal * invDelta * invRadiusGlobal * cscTheta * pDpx * 2 * ( 2 * div - invRadiusGlobal * pvpy);
    fReal f5 = reGlobal * invRadiusGlobal * cscTheta * (psspy + pspx + cosTheta * sigma12);

    // fReal f7 = 0.0; 		// Van der Waals
    fReal uAir = 0.0;
    if (currentTimeGlobal < 5)
	uAir = 20.f * (M_hPI - gTheta) * expf(-10 * powf(fabsf(gTheta - M_hPI), 2.f)) * radiusGlobal * cosf(gPhi) / UGlobal;

    fReal f6 = CrGlobal * (uAir - u1);
    
    // output
    fReal result = (u1 + timeStepGlobal * (f2 + f3 + f4 + f5 + CrGlobal * uAir))
	/ (1.0 + (CrGlobal + vTheta * invRadiusGlobal * cotTheta) * timeStepGlobal);
    if (fabsf(result) < eps)
	result = 0.f;
    velPhiOutput[thetaId * pitch + phiId] = result;
}


__global__ void resetThickness(float2* weight) {
    int splitVal = nPhiGlobal / blockDim.x;
    int threadSequence = blockIdx.x % splitVal;
    int phiId = threadIdx.x + threadSequence * blockDim.x;
    int thetaId = blockIdx.x / splitVal;

    weight[thetaId * nPhiGlobal + phiId].x = 0.;
    weight[thetaId * nPhiGlobal + phiId].y = 0.;
}

// __global__ void applyforceThickness
// (fReal* thicknessOutput, fReal* thicknessInput, fReal* div, size_t pitch)
// {
//     // Index
//     int splitVal = nPhiGlobal / blockDim.x;
//     int threadSequence = blockIdx.x % splitVal;
//     int phiId = threadIdx.x + threadSequence * blockDim.x;
//     int thetaId = blockIdx.x / splitVal;

//     fReal delta = thicknessInput[thetaId * pitch + phiId];
//     fReal f = div[thetaId * nPhiGlobal + phiId];
//     // implicit
//     thicknessOutput[thetaId * pitch + phiId] = delta / (1 + timeStepGlobal * f);
// }

__global__ void applyforceParticles
(fReal* tempVal, fReal* value, fReal* coord, fReal* div, size_t numOfParticles) {
    int particleId = blockIdx.x * blockDim.x + threadIdx.x;

    if (particleId < numOfParticles) {
	fReal phiId = coord[2 * particleId];
	fReal thetaId = coord[2 * particleId + 1];

	fReal f = sampleCentered(div, phiId, thetaId, nPhiGlobal);

	tempVal[particleId] = value[particleId] / (1 + timeStepGlobal * f);
    }
}


__global__ void mapParticlesToThickness
(fReal* particleCoord, fReal* particleVal, float2* weight, size_t numParticles)
{
    // Index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int particleId = index >> 1; // (index / 2)
    int partition = index & 1;	 // (index % 2)

    if (particleId < numParticles) {
	fReal gPhiId = particleCoord[2 * particleId];
	fReal gThetaId = particleCoord[2 * particleId + 1];

	fReal gTheta = gThetaId * gridLenGlobal;

	fReal sinTheta = max(sinf(gTheta), eps);

	fReal gPhi = gPhiId * gridLenGlobal;

	size_t thetaId = static_cast<size_t>(floorf(gThetaId));

	fReal x1 = cosf(gPhi) * sinTheta; fReal y1 = sinf(gPhi) * sinTheta; fReal z1 = cosf(gTheta);

	fReal theta = (thetaId + 0.5) * gridLenGlobal;

	fReal phiRange = 0.5/sinTheta;
	int minPhiId = static_cast<int>(ceilf(gPhiId - phiRange));
	int maxPhiId = static_cast<int>(floorf(gPhiId + phiRange));

	fReal z2 = cosf(theta);
	fReal r = sinf(theta);
	fReal value = particleVal[particleId];

	int begin; int end;
	
	if (partition == 0) {
	    begin = minPhiId; end = static_cast<int>(gPhiId);
	} else {
	    begin = static_cast<int>(gPhiId); end = maxPhiId + 1;
	}
	    
	for (int phiId = begin; phiId < end; phiId++) {
	    fReal phi = phiId * gridLenGlobal;
	    fReal x2 = cosf(phi) * r; fReal y2 = sinf(phi) * r;

	    fReal dist2 = powf(fabsf(x1 - x2), 2.f) + powf(fabsf(y1 - y2), 2.f) + powf(fabsf(z1 - z2), 2.f);
	        
	    if (dist2 <= .25f) {
		fReal w = expf(-10*dist2);
		size_t normalizedPhiId = (phiId + nPhiGlobal) % nPhiGlobal;
		float2* currentWeight = weight + (thetaId * nPhiGlobal + normalizedPhiId);
		atomicAdd(&(currentWeight->x), w);
		atomicAdd(&(currentWeight->y), w * value);
	    }
	}
    }
}


__global__ void normalizeThickness
(fReal* thicknessOutput, fReal* thicknessInput, fReal* velPhi, fReal* velTheta,
 fReal* div, float2* weight, size_t pitch) {
    // Index
    int splitVal = nPhiGlobal / blockDim.x;
    int threadSequence = blockIdx.x % splitVal;
    int phiId = threadIdx.x + threadSequence * blockDim.x;
    int thetaId = blockIdx.x / splitVal;

    float2* currentWeight = weight + (thetaId * nPhiGlobal + phiId);
    fReal w = currentWeight->x;
    fReal val = currentWeight->y;

    if (w > 0) {
	thicknessOutput[thetaId * pitch + phiId] = val / w;
    } else {
	//	printf("Warning: no particles contributed to grid thetaId %d, phiId %d\n", thetaId, phiId);
	fReal gPhiId = (fReal)phiId + centeredPhiOffset;
	fReal gThetaId = (fReal)thetaId + centeredThetaOffset;
	fReal gTheta = gThetaId * gridLenGlobal;

	// Sample the speed
	fReal guPhi = sampleVPhi(velPhi, gPhiId, gThetaId, pitch);
	fReal guTheta = sampleVTheta(velTheta, gPhiId, gThetaId, pitch);
    
	fReal cofTheta = timeStepGlobal * invRadiusGlobal * invGridLenGlobal;
	fReal sinTheta = max(sinf(gTheta), eps);
	fReal cofPhi = cofTheta / sinTheta;

	fReal deltaPhi = guPhi * cofPhi;
	fReal deltaTheta = guTheta * cofTheta;
	
# ifdef RUNGE_KUTTA
	// Traced halfway in phi-theta space
	fReal midPhiId = gPhiId - 0.5 * deltaPhi;
	fReal midThetaId = gThetaId - 0.5 * deltaTheta;
    
	fReal muPhi = sampleVPhi(velPhi, midPhiId, midThetaId, pitch);
	fReal muTheta = sampleVTheta(velTheta, midPhiId, midThetaId, pitch);

	deltaPhi = muPhi * cofPhi;
	deltaTheta = muTheta * cofTheta;
# endif

	fReal pPhiId = gPhiId - deltaPhi;
	fReal pThetaId = gThetaId - deltaTheta;

        fReal advectedVal = sampleCentered(thicknessInput, pPhiId, pThetaId, pitch);
	fReal f = div[thetaId * nPhiGlobal + phiId];
	thicknessOutput[thetaId * pitch + phiId] = advectedVal / (1 + timeStepGlobal * f);
    }
}


__global__ void applyforceSurfConcentration
(fReal* sConcentrationOutput, fReal* sConcentrationInput, fReal* div, size_t pitch)
{
    // Index
    int splitVal = nPhiGlobal / blockDim.x;
    int threadSequence = blockIdx.x % splitVal;
    int phiId = threadIdx.x + threadSequence * blockDim.x;
    int thetaId = blockIdx.x / splitVal;
	
    fReal gamma = sConcentrationInput[thetaId * pitch + phiId];
    fReal gammaWest = sConcentrationInput[thetaId * pitch + (phiId - 1 + nPhiGlobal) % nPhiGlobal];
    fReal gammaEast = sConcentrationInput[thetaId * pitch + (phiId + 1) % nPhiGlobal];
    fReal gammaNorth = 0.0;
    fReal gammaSouth = 0.0;
    if (thetaId != 0) {
    	gammaNorth = sConcentrationInput[(thetaId - 1) * pitch + phiId];
    } else {
    	size_t oppositePhiId = (phiId + nPhiGlobal / 2) % nPhiGlobal;
    	gammaNorth = sConcentrationInput[thetaId * pitch + oppositePhiId];
    }
    if (thetaId != nThetaGlobal - 1) {
    	gammaSouth = sConcentrationInput[(thetaId + 1) * pitch + phiId];
    } else {
    	size_t oppositePhiId = (phiId + nPhiGlobal / 2) % nPhiGlobal;
    	gammaSouth = sConcentrationInput[thetaId * pitch + oppositePhiId];
    }
    fReal laplace = invGridLenGlobal * invGridLenGlobal *
    	(gammaWest - 4*gamma + gammaEast + gammaNorth + gammaSouth);
    
    fReal f = div[thetaId * nPhiGlobal + phiId];
    fReal f2 = DsGlobal * laplace;

    sConcentrationOutput[thetaId * pitch + phiId] = max((gamma + f2 * timeStepGlobal) / (1 + timeStepGlobal * f), 0.f);
}


void KaminoSolver::bodyforce()
{
    dim3 gridLayout;
    dim3 blockLayout;

    bool noVis = false;
    for (int i = 0; i < 3; i++) {
    	determineLayout(gridLayout, blockLayout, velTheta->getNTheta(), velTheta->getNPhi());

	if (noVis) {
	    if (i == 0) {
		applyforcevelthetaNoViscosityKernel<<<gridLayout, blockLayout>>>
		    (velTheta->getGPUNextStep(), velTheta->getGPUThisStep(), velPhi->getGPUThisStep(),
		     thickness->getGPUThisStep(), surfConcentration->getGPUThisStep(),
		     velTheta->getNextStepPitchInElements());
	    } else {
		applyforcevelthetaNoViscosityKernel<<<gridLayout, blockLayout>>>
		    (velTheta->getGPUNextStep(), velTheta->getGPUThisStep(), velPhi->getGPUNextStep(),
		     thickness->getGPUNextStep(), surfConcentration->getGPUNextStep(),
		     velTheta->getNextStepPitchInElements()); 
	    }
	} else {
	    if (i == 0) {
		applyforcevelthetaKernel<<<gridLayout, blockLayout>>>
		    (velTheta->getGPUNextStep(), velTheta->getGPUThisStep(), velPhi->getGPUThisStep(),
		     thickness->getGPUThisStep(), surfConcentration->getGPUThisStep(), div,
		     velTheta->getNextStepPitchInElements());
	    } else {    	
		applyforcevelthetaKernel<<<gridLayout, blockLayout>>>
		    (velTheta->getGPUNextStep(), velTheta->getGPUThisStep(), velPhi->getGPUNextStep(),
		     thickness->getGPUNextStep(), surfConcentration->getGPUNextStep(), div,
		     velTheta->getNextStepPitchInElements());
	    }
	}

    	checkCudaErrors(hipGetLastError());
    	checkCudaErrors(hipDeviceSynchronize());

	determineLayout(gridLayout, blockLayout, velPhi->getNTheta(), velPhi->getNPhi());
	if (noVis) {
	    if (i == 0) {
		applyforcevelphiNoViscosityKernel<<<gridLayout, blockLayout>>>
		    (velPhi->getGPUNextStep(), velTheta->getGPUThisStep(), velPhi->getGPUThisStep(),
		     thickness->getGPUThisStep(), surfConcentration->getGPUThisStep(),
		     velPhi->getNextStepPitchInElements());
	    } else {    	
		applyforcevelphiNoViscosityKernel<<<gridLayout, blockLayout>>>
		    (velPhi->getGPUNextStep(), velTheta->getGPUNextStep(), velPhi->getGPUThisStep(),
		     thickness->getGPUNextStep(), surfConcentration->getGPUNextStep(),
		     velPhi->getNextStepPitchInElements());
	    }
	} else {
	    if (i == 0) {
		applyforcevelphiKernel<<<gridLayout, blockLayout>>>
		    (velPhi->getGPUNextStep(), velTheta->getGPUThisStep(), velPhi->getGPUThisStep(),
		     thickness->getGPUThisStep(), surfConcentration->getGPUThisStep(), div,
		     velPhi->getNextStepPitchInElements());
	    } else {
		applyforcevelphiKernel<<<gridLayout, blockLayout>>>
		    (velPhi->getGPUNextStep(), velTheta->getGPUNextStep(), velPhi->getGPUThisStep(),
		     thickness->getGPUNextStep(), surfConcentration->getGPUNextStep(), div,
		     velPhi->getNextStepPitchInElements());
	    }
	}
	
    	checkCudaErrors(hipGetLastError());
    	checkCudaErrors(hipDeviceSynchronize());

	determineLayout(gridLayout, blockLayout, nTheta, nPhi);
	comDivergenceKernel<<<gridLayout, blockLayout>>>
	    (div, velPhi->getGPUNextStep(), velTheta->getGPUNextStep(),
	     velPhi->getNextStepPitchInElements(), velTheta->getNextStepPitchInElements());
	checkCudaErrors(hipGetLastError());

	determineLayout(gridLayout, blockLayout, nTheta, nPhi);
	resetThickness<<<gridLayout, blockLayout>>>(weight);
	checkCudaErrors(hipGetLastError());
    
	if (particles->numOfParticles > 0) {
	    determineLayout(gridLayout, blockLayout, 1, particles->numOfParticles);
	    applyforceParticles<<<gridLayout, blockLayout>>>
		(particles->tempVal, particles->value, particles->coordGPUThisStep, div, particles->numOfParticles);
	    checkCudaErrors(hipGetLastError());
	    checkCudaErrors(hipDeviceSynchronize());

	    determineLayout(gridLayout, blockLayout, 2, particles->numOfParticles);
	    mapParticlesToThickness<<<gridLayout, blockLayout>>>
		(particles->coordGPUThisStep, particles->tempVal,  weight, particles->numOfParticles);
	    checkCudaErrors(hipGetLastError());
	}

	determineLayout(gridLayout, blockLayout, surfConcentration->getNTheta(), surfConcentration->getNPhi());
	applyforceSurfConcentration<<<gridLayout, blockLayout>>>
	    (surfConcentration->getGPUNextStep(), surfConcentration->getGPUThisStep(),
	     div, surfConcentration->getNextStepPitchInElements());
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
        
	determineLayout(gridLayout, blockLayout, thickness->getNTheta(), thickness->getNPhi());
	if (i == 0) {
	    normalizeThickness<<<gridLayout, blockLayout>>>
		(thickness->getGPUNextStep(), thickness->getGPUThisStep(), velPhi->getGPUThisStep(),
		 velTheta->getGPUThisStep(), div, weight, thickness->getNextStepPitchInElements());
	} else {
	    normalizeThickness<<<gridLayout, blockLayout>>>
		(thickness->getGPUNextStep(), thickness->getGPUThisStep(), velPhi->getGPUNextStep(),
		 velTheta->getGPUNextStep(), div, weight, thickness->getNextStepPitchInElements());
	}
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());       
    }
 
    std::swap(particles->tempVal, particles->value);
    thickness->swapGPUBuffer();
    surfConcentration->swapGPUBuffer();
    swapVelocityBuffers();
}

__global__ void fillDivergenceKernel
(ComplexFourier* outputF, fReal* velPhi, fReal* velTheta,
 size_t velPhiPitchInElements, size_t velThetaPitchInElements)
{
    int splitVal = nPhiGlobal / blockDim.x;
    int threadSequence = blockIdx.x % splitVal;
    int gridPhiId = threadIdx.x + threadSequence * blockDim.x;
    int gridThetaId = blockIdx.x / splitVal;
    //fReal gridPhiCoord = ((fReal)gridPhiId + centeredPhiOffset) * gridLen;
    fReal gridThetaCoord = ((fReal)gridThetaId + centeredThetaOffset) * gridLenGlobal;

    fReal uEast = 0.0;
    fReal uWest = 0.0;
    fReal vNorth = 0.0;
    fReal vSouth = 0.0;

    fReal halfStep = 0.5 * gridLenGlobal;

    fReal thetaSouth = gridThetaCoord + halfStep;
    fReal thetaNorth = gridThetaCoord - halfStep;

    int phiIdWest = gridPhiId;
    int phiIdEast = (phiIdWest + 1) % nPhiGlobal;

    uWest = velPhi[gridThetaId * velPhiPitchInElements + phiIdWest];
    uEast = velPhi[gridThetaId * velPhiPitchInElements + phiIdEast];

    if (gridThetaId != 0)
	{
	    int thetaNorthIdx = gridThetaId - 1;
	    vNorth = velTheta[thetaNorthIdx * velThetaPitchInElements + gridPhiId];
	}
    if (gridThetaId != nThetaGlobal - 1)
	{
	    int thetaSouthIdx = gridThetaId;
	    vSouth = velTheta[thetaSouthIdx * velThetaPitchInElements + gridPhiId];
	}

    fReal invGridSine = 1.0 / sinf(gridThetaCoord);
    fReal sinNorth = sinf(thetaNorth);
    fReal sinSouth = sinf(thetaSouth);
    fReal factor = invGridSine / gridLenGlobal;
    fReal termTheta = factor * (vSouth * sinSouth - vNorth * sinNorth);
    fReal termPhi = factor * (uEast - uWest);

    fReal div = termTheta + termPhi;

    ComplexFourier f;
    f.x = div;
    f.y = 0.0;
    outputF[gridThetaId * nPhiGlobal + gridPhiId] = f;
}

__global__ void shiftFKernel
(ComplexFourier* FFourierInput, fReal* FFourierShiftedReal, fReal* FFourierShiftedImag)
{
    int splitVal = nPhiGlobal / blockDim.x;
    int threadSequence = blockIdx.x % splitVal;
    int nIdx = threadIdx.x + threadSequence * blockDim.x;
    int thetaIdx = blockIdx.x / splitVal;

    int fftIndex = nPhiGlobal / 2 - nIdx;
    if (fftIndex < 0)
	fftIndex += nPhiGlobal;
    //FFourierShifted[thetaIdx * nPhi + phiIdx] = FFourierInput[thetaIdx * nPhi + fftIndex];
    fReal real = FFourierInput[thetaIdx * nPhiGlobal + fftIndex].x / (fReal)nPhiGlobal;
    fReal imag = FFourierInput[thetaIdx * nPhiGlobal + fftIndex].y / (fReal)nPhiGlobal;
    FFourierShiftedReal[nIdx * nThetaGlobal + thetaIdx] = real;
    FFourierShiftedImag[nIdx * nThetaGlobal + thetaIdx] = imag;
}

__global__ void copy2UFourier
(ComplexFourier* UFourierOutput, fReal* UFourierReal, fReal* UFourierImag)
{
    int splitVal = nPhiGlobal / blockDim.x;
    int threadSequence = blockIdx.x % splitVal;
    int nIdx = threadIdx.x + threadSequence * blockDim.x;
    int thetaIdx = blockIdx.x / splitVal;

    ComplexFourier u;
    u.x = UFourierReal[nIdx * nThetaGlobal + thetaIdx];
    u.y = UFourierImag[nIdx * nThetaGlobal + thetaIdx];
    UFourierOutput[thetaIdx * nPhiGlobal + nIdx] = u;
}

__global__ void cacheZeroComponents
(fReal* zeroComponentCache, ComplexFourier* input)
{
    int splitVal = nThetaGlobal / blockDim.x;
    int threadSequence = blockIdx.x % splitVal;
    int thetaIdx = threadIdx.x + threadSequence * blockDim.x;

    zeroComponentCache[thetaIdx] = input[thetaIdx * nPhiGlobal + nPhiGlobal / 2].x;
}

__global__ void shiftUKernel
(ComplexFourier* UFourierInput, fReal* pressure, fReal* zeroComponentCache,
 size_t nPressurePitchInElements)
{
    int splitVal = nPhiGlobal / blockDim.x;
    int threadSequence = blockIdx.x % splitVal;
    int phiIdx = threadIdx.x + threadSequence * blockDim.x;
    int thetaIdx = blockIdx.x / splitVal;

    int fftIndex = 0;
    fReal zeroComponent = zeroComponentCache[thetaIdx];
    if (phiIdx != 0)
	fftIndex = nPhiGlobal - phiIdx;
    fReal pressureVal;

    if (phiIdx % 2 == 0)
	pressureVal = UFourierInput[thetaIdx * nPhiGlobal + fftIndex].x - zeroComponent;
    else
	pressureVal = -UFourierInput[thetaIdx * nPhiGlobal + fftIndex].x - zeroComponent;

    pressure[thetaIdx * nPressurePitchInElements + phiIdx] = pressureVal;
}

__global__ void applyPressureTheta
(fReal* output, fReal* prev, fReal* pressure,
 size_t nPitchInElementsPressure, size_t nPitchInElementsVTheta)
{
    int splitVal = nPhiGlobal / blockDim.x;
    int threadSequence = blockIdx.x % splitVal;
    int phiId = threadIdx.x + threadSequence * blockDim.x;
    int thetaId = blockIdx.x / splitVal;

    int pressureThetaNorthId = thetaId;
    int pressureThetaSouthId = thetaId + 1;
    fReal pressureNorth = pressure[pressureThetaNorthId * nPitchInElementsPressure + phiId];
    fReal pressureSouth = pressure[pressureThetaSouthId * nPitchInElementsPressure + phiId];

    fReal deltaVTheta = (pressureSouth - pressureNorth) / (-gridLenGlobal);
    fReal previousVTheta = prev[thetaId * nPitchInElementsVTheta + phiId];
    output[thetaId * nPitchInElementsVTheta + phiId] = previousVTheta + deltaVTheta;
}

__global__ void applyPressurePhi
(fReal* output, fReal* prev, fReal* pressure,
 size_t nPitchInElementsPressure, size_t nPitchInElementsVPhi)
{
    int splitVal = nPhiGlobal / blockDim.x;
    int threadSequence = blockIdx.x % splitVal;
    int phiId = threadIdx.x + threadSequence * blockDim.x;
    int thetaId = blockIdx.x / splitVal;

    int pressurePhiWestId;
    if (phiId == 0)
	pressurePhiWestId = nPhiGlobal - 1;
    else
	pressurePhiWestId = phiId - 1;
    int pressurePhiEastId = phiId;

    fReal pressureWest = pressure[thetaId * nPitchInElementsPressure + pressurePhiWestId];
    fReal pressureEast = pressure[thetaId * nPitchInElementsPressure + pressurePhiEastId];

    fReal thetaBelt = (thetaId + centeredThetaOffset) * gridLenGlobal;
    fReal deltaVPhi = (pressureEast - pressureWest) / (-gridLenGlobal * sinf(thetaBelt));
    fReal previousVPhi = prev[thetaId * nPitchInElementsVPhi + phiId];
    output[thetaId * nPitchInElementsVPhi + phiId] = previousVPhi + deltaVPhi;
}

void KaminoSolver::projection()
{
    dim3 gridLayout;
    dim3 blockLayout;
    determineLayout(gridLayout, blockLayout, nTheta, nPhi);
    fillDivergenceKernel<<<gridLayout, blockLayout>>>
	(this->gpuFFourier, this->velPhi->getGPUThisStep(), this->velTheta->getGPUThisStep(),
	 this->velPhi->getThisStepPitchInElements(), this->velTheta->getThisStepPitchInElements());
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());



    // Note that cuFFT inverse returns results are SigLen times larger
    checkCudaErrors((hipError_t)hipfftExecC2C(this->kaminoPlan,
					      this->gpuFFourier, this->gpuFFourier, HIPFFT_BACKWARD));
    checkCudaErrors(hipGetLastError());



    // Siglen is nPhi
    determineLayout(gridLayout, blockLayout, nTheta, nPhi);
    shiftFKernel<<<gridLayout, blockLayout>>>
	(gpuFFourier, gpuFReal, gpuFImag);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    // Now gpuFDivergence stores all the Fn



    gridLayout = dim3(nPhi);
    blockLayout = dim3(nTheta / 2);
    const unsigned sharedMemSize = nTheta * 5 * sizeof(fReal);
    crKernel<<<gridLayout, blockLayout, sharedMemSize>>>
	(this->gpuA, this->gpuB, this->gpuC, this->gpuFReal, this->gpuUReal);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    gridLayout = dim3(nPhi);
    blockLayout = dim3(nTheta / 2);
    crKernel<<<gridLayout, blockLayout, sharedMemSize>>>
	(this->gpuA, this->gpuB, this->gpuC, this->gpuFImag, this->gpuUImag);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());



    determineLayout(gridLayout, blockLayout, nTheta, nPhi);
    copy2UFourier<<<gridLayout, blockLayout>>>
	(this->gpuUFourier, this->gpuUReal, this->gpuUImag);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());



    determineLayout(gridLayout, blockLayout, 1, nTheta);
    cacheZeroComponents<<<gridLayout, blockLayout>>>
	(gpuFZeroComponent, gpuUFourier);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());



    checkCudaErrors((hipError_t)hipfftExecC2C(this->kaminoPlan,
					      this->gpuUFourier, this->gpuUFourier, HIPFFT_FORWARD));
    checkCudaErrors(hipGetLastError());



    determineLayout(gridLayout, blockLayout, nTheta, nPhi);
    shiftUKernel<<<gridLayout, blockLayout>>>
	(gpuUFourier, pressure->getGPUThisStep(), this->gpuFZeroComponent,
	 pressure->getThisStepPitchInElements());
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    //pressure->copyBackToCPU();

    determineLayout(gridLayout, blockLayout, velTheta->getNTheta(), velTheta->getNPhi());
    applyPressureTheta<<<gridLayout, blockLayout>>>
	(velTheta->getGPUNextStep(), velTheta->getGPUThisStep(), pressure->getGPUThisStep(),
	 pressure->getThisStepPitchInElements(), velTheta->getNextStepPitchInElements());
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    determineLayout(gridLayout, blockLayout, velPhi->getNTheta(), velPhi->getNPhi());
    applyPressurePhi<<<gridLayout, blockLayout>>>
	(velPhi->getGPUNextStep(), velPhi->getGPUThisStep(), pressure->getGPUThisStep(),
	 pressure->getThisStepPitchInElements(), velPhi->getNextStepPitchInElements());
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    swapVelocityBuffers();
}


Kamino::Kamino(fReal radius, fReal H, fReal U, fReal c_m, fReal Gamma_m,
	       fReal T, fReal Ds, fReal rm, size_t nTheta, 
	       float dt, float DT, int frames, fReal A, int B, int C, int D, int E,
	       std::string thicknessPath, std::string velocityPath,
	       std::string thicknessImage, size_t particleDensity, int device):
    radius(radius), invRadius(1/radius), H(H), U(U), c_m(c_m), Gamma_m(Gamma_m), T(T),
    Ds(Ds), gs(g/(U*U)), rm(rm), epsilon(H), sigma_r(R*T), M(Gamma_m*R*T/(3*rho*H*U*U)),
    S(sigma_a*H/(2*mu*U)), re(mu/(rho*U)), Cr(rhoa*sqrt(mua)/(rho*U*H)),
    nTheta(nTheta), nPhi(2 * nTheta),
    gridLen(M_PI / nTheta), invGridLen(nTheta / M_PI), 
    dt(dt), DT(DT), frames(frames),
    A(A), B(B), C(C), D(D), E(E),
    thicknessPath(thicknessPath), velocityPath(velocityPath),
    thicknessImage(thicknessImage), particleDensity(particleDensity), device(device)
{
    std::cout << "Re^-1 " << re << std::endl;
    std::cout << "S " << S << std::endl;
    std::cout << "Cr " << Cr << std::endl;
    std::cout << "M " << M << std::endl;
}
Kamino::~Kamino()
{}

void Kamino::run()
{
    KaminoSolver solver(nPhi, nTheta, radius, dt, A, B, C, D, E, H, device);

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(nPhiGlobal), &(this->nPhi), sizeof(size_t)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(nThetaGlobal), &(this->nTheta), sizeof(size_t)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(radiusGlobal), &(this->radius), sizeof(fReal)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(invRadiusGlobal), &(this->invRadius), sizeof(fReal)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(timeStepGlobal), &(this->dt), sizeof(fReal)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(gridLenGlobal), &(this->gridLen), sizeof(fReal)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(invGridLenGlobal), &(this->invGridLen), sizeof(fReal)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(SGlobal), &(this->S), sizeof(fReal)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(MGlobal), &(this->M), sizeof(fReal)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(reGlobal), &(this->re), sizeof(fReal)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(gGlobal), &(this->gs), sizeof(fReal)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(DsGlobal), &(this->Ds), sizeof(fReal)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(CrGlobal), &(this->Cr), sizeof(fReal)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(UGlobal), &(this->U), sizeof(fReal)));

    solver.initThicknessfromPic(thicknessImage, this->particleDensity);

# ifdef WRITE_THICKNESS_DATA
    solver.write_thickness_img(thicknessPath, 0);
# endif  
# ifdef WRITE_VELOCITY_DATA
    solver.write_velocity_image(velocityPath, 0);
    size_t split = velocityPath.find("/");
    std::string concentrationPath = velocityPath;
    concentrationPath.replace(concentrationPath.begin() + split + 1, concentrationPath.end(), "con");
    solver.write_concentration_image(concentrationPath, 0);
# endif

# ifdef PERFORMANCE_BENCHMARK
    KaminoTimer timer;
    timer.startTimer();
# endif

    float T = 0.0;              // simulation time
    int i = 1;
    for (; i < frames; i++) {
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(currentTimeGlobal), &T, sizeof(fReal)));
	std::cout << "current time " << T << std::endl;
	// if (i > 1)
	//     solver.adjustStepSize(dt, U, epsilon);
	//dt = 0.0005;

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(timeStepGlobal), &dt, sizeof(fReal)));
	std::cout << "current time step size is " << dt << " s" << std::endl;
	std::cout << "steps needed until next frame " << DT/dt*U << std::endl;
    
	while ((T + dt/this->U) <= i*DT && !solver.isBroken()) {
	    solver.stepForward();
	    T += dt/this->U;
	}
	if (T < i*DT) {
	    solver.stepForward(i*DT - T);
	}
	if (solver.isBroken()) {
	    std::cerr << "Film is broken." << std::endl;
	    break;
	}
	T = i*DT;

	std::cout << "Frame " << i << " is ready" << std::endl;
# ifdef WRITE_THICKNESS_DATA
	solver.write_thickness_img(thicknessPath, i);
# endif
# ifdef WRITE_VELOCITY_DATA
	solver.write_velocity_image(velocityPath, i);
	solver.write_concentration_image(concentrationPath, i);
# endif
    }

# ifdef PERFORMANCE_BENCHMARK
    float gpu_time = timer.stopTimer();
# endif

    std::cout << "Time spent: " << gpu_time << "ms" << std::endl;
    std::cout << "Performance: " << 1000.0 * i / gpu_time << " frames per second" << std::endl;
}
