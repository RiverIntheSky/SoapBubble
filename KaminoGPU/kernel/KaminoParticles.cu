#include "hip/hip_runtime.h"
# include "../include/KaminoParticles.cuh"

KaminoParticles::KaminoParticles(std::string path, size_t particleDensity, float gridLen, size_t nTheta)
    : particlePGrid(particlePGrid), nPhi(2 * nTheta), nTheta(nTheta)
{
    numOfParticles = nTheta * nPhi * particleDensity;

    checkCudaErrors(hipMalloc(&coordGPUThisStep, sizeof(float) * numOfParticles * 2));
    checkCudaErrors(hipMalloc(&coordGPUNextStep, sizeof(float) * numOfParticles * 2));
    coordCPUBuffer = new float[numOfParticles * 2];

    for (size_t i = 0; i < numOfParticles; i++) {
	float u1 = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
	float u2 = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
	float phiId = u1 * nPhi;
	//float thetaId = u2 * nTheta;
        float thetaId = acosf(1 - 2 * u2) * nTheta / M_PI;
	coordCPUBuffer[2 * i] = thetaId;
	coordCPUBuffer[2 * i + 1] = phiId;
    }

    copy2GPU();
    checkCudaErrors(hipMalloc(&value, sizeof(float) * numOfParticles));
    checkCudaErrors(hipMalloc(&tempVal, sizeof(float) * numOfParticles));
}

KaminoParticles::~KaminoParticles()
{
    delete[] coordCPUBuffer;
    delete[] value;
    delete[] tempVal;

    checkCudaErrors(hipFree(coordGPUThisStep));
    checkCudaErrors(hipFree(coordGPUNextStep));
}

void KaminoParticles::copy2GPU()
{
    if (numOfParticles != 0)
	{
	    checkCudaErrors(hipMemcpy(this->coordGPUThisStep, this->coordCPUBuffer,
				       sizeof(float) * numOfParticles * 2, hipMemcpyHostToDevice));
	}
}

void KaminoParticles::copyBack2CPU()
{
    if (numOfParticles != 0)
	{
	    checkCudaErrors(hipMemcpy(this->coordCPUBuffer, this->coordGPUThisStep,
				       sizeof(float) * numOfParticles * 2, hipMemcpyDeviceToHost));
	}
}

void KaminoParticles::swapGPUBuffers()
{
    float* temp = this->coordGPUThisStep;
    this->coordGPUThisStep = this->coordGPUNextStep;
    this->coordGPUNextStep = temp;
}