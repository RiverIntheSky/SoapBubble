# include "../include/timer.cuh"

Timer::Timer() : timeElapsed(0.0f)
{
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
}
Timer::~Timer()
{
    hipEventDestroy(start);
    hipEventDestroy(stop);
}
void Timer::startTimer()
{
    hipEventRecord(start, 0);
}
float Timer::stopTimer()
{
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    checkCudaErrors(hipEventElapsedTime(&timeElapsed, start, stop));
    return timeElapsed;
}