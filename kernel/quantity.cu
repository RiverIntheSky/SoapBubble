# include "../include/quantity.cuh"

void Quantity::copyToGPU() {
    /* 
       Pitch : nPhi * sizeof(fReal)
       Width : nPhi * sizeof(fReal)
       Height: nTheta
    */
    checkCudaErrors(hipMemcpy2D(gpuThisStep, thisStepPitch, cpuBuffer, 
				 nPhi * sizeof(fReal), nPhi * sizeof(fReal), nTheta, hipMemcpyHostToDevice));
}


void Quantity::copyBackToCPU() {
    checkCudaErrors(hipMemcpy2D((void*)this->cpuBuffer, nPhi * sizeof(fReal), (void*)this->gpuThisStep,
				 this->thisStepPitch, nPhi * sizeof(fReal), nTheta, hipMemcpyDeviceToHost));
}


Quantity::Quantity(std::string attributeName, size_t nPhi, size_t nTheta,
			       fReal phiOffset, fReal thetaOffset)
    : nPhi(nPhi), nTheta(nTheta), gridLen(M_2PI / nPhi), invGridLen(1.0 / gridLen),
    attrName(attributeName), phiOffset(phiOffset), thetaOffset(thetaOffset) {
    cpuBuffer = new fReal[nPhi * nTheta];
    checkCudaErrors(hipMallocPitch(&gpuThisStep, &thisStepPitch, nPhi * sizeof(fReal), nTheta));
    checkCudaErrors(hipMallocPitch(&gpuNextStep, &nextStepPitch, nPhi * sizeof(fReal), nTheta));
    checkCudaErrors(hipMemset(gpuThisStep, 0, thisStepPitch * nTheta));
    checkCudaErrors(hipMemset(gpuNextStep, 0, nextStepPitch * nTheta));
}


BimocqQuantity::BimocqQuantity(std::string attributeName, size_t nPhi, size_t nTheta,
			       fReal phiOffset, fReal thetaOffset)
    : Quantity(attributeName, nPhi, nTheta, phiOffset, thetaOffset) {
    checkCudaErrors(hipMallocPitch(&gpuInit, &thisStepPitch, nPhi * sizeof(fReal), nTheta));
    checkCudaErrors(hipMallocPitch(&gpuDelta, &thisStepPitch, nPhi * sizeof(fReal), nTheta));
    checkCudaErrors(hipMallocPitch(&gpuInitLast, &thisStepPitch, nPhi * sizeof(fReal), nTheta));
    checkCudaErrors(hipMallocPitch(&gpuDeltaLast, &thisStepPitch, nPhi * sizeof(fReal), nTheta));
    checkCudaErrors(hipMemset(gpuInit, 0, thisStepPitch * nTheta));
    checkCudaErrors(hipMemset(gpuDelta, 0, thisStepPitch * nTheta));
    checkCudaErrors(hipMemset(gpuInitLast, 0, thisStepPitch * nTheta));
    checkCudaErrors(hipMemset(gpuDeltaLast, 0, thisStepPitch * nTheta));
}


Quantity::~Quantity() {
    delete[] cpuBuffer;

    checkCudaErrors(hipFree(gpuThisStep));
    checkCudaErrors(hipFree(gpuNextStep));
}


BimocqQuantity::~BimocqQuantity() {
    checkCudaErrors(hipFree(gpuInit));
    checkCudaErrors(hipFree(gpuDelta));
    checkCudaErrors(hipFree(gpuInitLast));
    checkCudaErrors(hipFree(gpuDeltaLast));
}


std::string Quantity::getName() {
    return this->attrName;
}


size_t Quantity::getNPhi() {
    return this->nPhi;
}


size_t Quantity::getNTheta() {
    return this->nTheta;
}


void Quantity::swapGPUBuffer() {
    fReal* tempPtr = this->gpuThisStep;
    this->gpuThisStep = this->gpuNextStep;
    this->gpuNextStep = tempPtr;
}


fReal Quantity::getCPUValueAt(size_t phi, size_t theta) {
    return this->accessCPUValueAt(phi, theta);
}


void Quantity::setCPUValueAt(size_t phi, size_t theta, fReal val) {
    this->accessCPUValueAt(phi, theta) = val;
}


fReal& Quantity::accessCPUValueAt(size_t phi, size_t theta) {
    assert(theta < nTheta && phi < nPhi);
    return this->cpuBuffer[theta * nPhi + phi];
}


fReal Quantity::getThetaOffset() {
    return this->thetaOffset;
}


fReal Quantity::getPhiOffset() {
    return this->phiOffset;
}


fReal* Quantity::getGPUThisStep() {
    return this->gpuThisStep;
}


fReal* Quantity::getGPUNextStep() {
    return this->gpuNextStep;
}


fReal*& BimocqQuantity::getGPUInit() {
    return this->gpuInit;
}


fReal*& BimocqQuantity::getGPUDelta() {
    return this->gpuDelta;
}


fReal*& BimocqQuantity::getGPUInitLast() {
    return this->gpuInitLast;
}


fReal*& BimocqQuantity::getGPUDeltaLast() {
    return this->gpuDeltaLast;
}


size_t Quantity::getThisStepPitchInElements() {
    return this->thisStepPitch / sizeof(fReal);
}


size_t Quantity::getNextStepPitchInElements() {
    return this->nextStepPitch / sizeof(fReal);
}